#include "hip/hip_runtime.h"
/*

                 Copyright (c) 2010.
      Lawrence Livermore National Security, LLC.
Produced at the Lawrence Livermore National Laboratory.
                  LLNL-CODE-461231
                All rights reserved.

This file is part of LULESH, Version 1.0.
Please also read this link -- http://www.opensource.org/licenses/index.php

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

   * Redistributions of source code must retain the above copyright
     notice, this list of conditions and the disclaimer below.

   * Redistributions in binary form must reproduce the above copyright
     notice, this list of conditions and the disclaimer (as noted below)
     in the documentation and/or other materials provided with the
     distribution.

   * Neither the name of the LLNS/LLNL nor the names of its contributors
     may be used to endorse or promote products derived from this software
     without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
ARE DISCLAIMED. IN NO EVENT SHALL LAWRENCE LIVERMORE NATIONAL SECURITY, LLC,
THE U.S. DEPARTMENT OF ENERGY OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.


Additional BSD Notice

1. This notice is required to be provided under our contract with the U.S.
   Department of Energy (DOE). This work was produced at Lawrence Livermore
   National Laboratory under Contract No. DE-AC52-07NA27344 with the DOE.

2. Neither the United States Government nor Lawrence Livermore National
   Security, LLC nor any of their employees, makes any warranty, express
   or implied, or assumes any liability or responsibility for the accuracy,
   completeness, or usefulness of any information, apparatus, product, or
   process disclosed, or represents that its use would not infringe
   privately-owned rights.

3. Also, reference herein to any specific commercial products, process, or
   services by trade name, trademark, manufacturer or otherwise does not
   necessarily constitute or imply its endorsement, recommendation, or
   favoring by the United States Government or Lawrence Livermore National
   Security, LLC. The views and opinions of authors expressed herein do not
   necessarily state or reflect those of the United States Government or
   Lawrence Livermore National Security, LLC, and shall not be used for
   advertising or product endorsement purposes.

*/

#include "../test/alpaka_vector_test.h"
#include "allocator.h"
#include "alpaka_utils.h"
#include "hip/hip_runtime_api.h"
#include "lulesh_kernels.h"
#include "sm_utils.inl"
#include "util.h"

#include <alpaka/alpaka.hpp>

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include <fstream>
#include <iomanip>
#include <iostream>
#include <sstream>

/*
#ifdef USE_MPI
#include <mpi.h>
#endif
*/
#include "lulesh.h"

#include <sys/time.h>
#include <unistd.h>
#define TEST
#define ALPAKA

/****************************************************/
/* Allow flexibility for arithmetic representations */
/****************************************************/

inline __device__ __host__ real4 FABS(real4 arg)
{
    return fabsf(arg);
}

inline __device__ __host__ real8 FABS(real8 arg)
{
    return fabs(arg);
}

template<typename T>
T SQRT(T x)
{
    return alpaka::math::sqrt(x);
};

#define MAX(a, b) (((a) > (b)) ? (a) : (b))

template<typename T>
T FMAX(T x, T y)
{
    return MAX(x, y);
};

/* Stuff needed for boundary conditions */
/* 2 BCs on each of 6 hexahedral faces (12 bits) */
#define XI_M 0x0'0007
#define XI_M_SYMM 0x0'0001
#define XI_M_FREE 0x0'0002
#define XI_M_COMM 0x0'0004

#define XI_P 0x0'0038
#define XI_P_SYMM 0x0'0008
#define XI_P_FREE 0x0'0010
#define XI_P_COMM 0x0'0020

#define ETA_M 0x0'01c0
#define ETA_M_SYMM 0x0'0040
#define ETA_M_FREE 0x0'0080
#define ETA_M_COMM 0x0'0100

#define ETA_P 0x0'0e00
#define ETA_P_SYMM 0x0'0200
#define ETA_P_FREE 0x0'0400
#define ETA_P_COMM 0x0'0800

#define ZETA_M 0x0'7000
#define ZETA_M_SYMM 0x0'1000
#define ZETA_M_FREE 0x0'2000
#define ZETA_M_COMM 0x0'4000

#define ZETA_P 0x3'8000
#define ZETA_P_SYMM 0x0'8000
#define ZETA_P_FREE 0x1'0000
#define ZETA_P_COMM 0x2'0000

#define VOLUDER(a0, a1, a2, a3, a4, a5, b0, b1, b2, b3, b4, b5, dvdc)                                                 \
    {                                                                                                                 \
        const Real_t twelfth = Real_t(1.0) / Real_t(12.0);                                                            \
                                                                                                                      \
        dvdc = ((a1) + (a2)) * ((b0) + (b1)) - ((a0) + (a1)) * ((b1) + (b2)) + ((a0) + (a4)) * ((b3) + (b4))          \
               - ((a3) + (a4)) * ((b0) + (b4)) - ((a2) + (a5)) * ((b3) + (b5)) + ((a3) + (a5)) * ((b2) + (b5));       \
        dvdc *= twelfth;                                                                                              \
    }

/*
__device__

__forceinline__
void SumOverNodes(Real_t& val, volatile Real_t* smem, int cta_elem, int node) {

  int tid = (cta_elem << 3) + node;
  smem[tid] = val;
  if (node < 4)
  {
    smem[tid] += smem[tid+4];
    smem[tid] += smem[tid+2];
    smem[tid] += smem[tid+1];
  }
  val = smem[(cta_elem << 3)];
}
*/

__device__ __forceinline__ void SumOverNodesShfl(Real_t& val)
{
    val += utils::shfl_xor(val, 4, 8);
    val += utils::shfl_xor(val, 2, 8);
    val += utils::shfl_xor(val, 1, 8);
}

__host__ __device__ __forceinline__ Real_t CalcElemVolume(
    Real_t const x0,
    Real_t const x1,
    Real_t const x2,
    Real_t const x3,
    Real_t const x4,
    Real_t const x5,
    Real_t const x6,
    Real_t const x7,
    Real_t const y0,
    Real_t const y1,
    Real_t const y2,
    Real_t const y3,
    Real_t const y4,
    Real_t const y5,
    Real_t const y6,
    Real_t const y7,
    Real_t const z0,
    Real_t const z1,
    Real_t const z2,
    Real_t const z3,
    Real_t const z4,
    Real_t const z5,
    Real_t const z6,
    Real_t const z7)
{
    Real_t twelveth = Real_t(1.0) / Real_t(12.0);

    Real_t dx61 = x6 - x1;
    Real_t dy61 = y6 - y1;
    Real_t dz61 = z6 - z1;

    Real_t dx70 = x7 - x0;
    Real_t dy70 = y7 - y0;
    Real_t dz70 = z7 - z0;

    Real_t dx63 = x6 - x3;
    Real_t dy63 = y6 - y3;
    Real_t dz63 = z6 - z3;

    Real_t dx20 = x2 - x0;
    Real_t dy20 = y2 - y0;
    Real_t dz20 = z2 - z0;

    Real_t dx50 = x5 - x0;
    Real_t dy50 = y5 - y0;
    Real_t dz50 = z5 - z0;

    Real_t dx64 = x6 - x4;
    Real_t dy64 = y6 - y4;
    Real_t dz64 = z6 - z4;

    Real_t dx31 = x3 - x1;
    Real_t dy31 = y3 - y1;
    Real_t dz31 = z3 - z1;

    Real_t dx72 = x7 - x2;
    Real_t dy72 = y7 - y2;
    Real_t dz72 = z7 - z2;

    Real_t dx43 = x4 - x3;
    Real_t dy43 = y4 - y3;
    Real_t dz43 = z4 - z3;

    Real_t dx57 = x5 - x7;
    Real_t dy57 = y5 - y7;
    Real_t dz57 = z5 - z7;

    Real_t dx14 = x1 - x4;
    Real_t dy14 = y1 - y4;
    Real_t dz14 = z1 - z4;

    Real_t dx25 = x2 - x5;
    Real_t dy25 = y2 - y5;
    Real_t dz25 = z2 - z5;

#define TRIPLE_PRODUCT(x1, y1, z1, x2, y2, z2, x3, y3, z3)                                                            \
    ((x1) * ((y2) * (z3) - (z2) * (y3)) + (x2) * ((z1) * (y3) - (y1) * (z3)) + (x3) * ((y1) * (z2) - (z1) * (y2)))

    // 11 + 3*14
    Real_t volume = TRIPLE_PRODUCT(dx31 + dx72, dx63, dx20, dy31 + dy72, dy63, dy20, dz31 + dz72, dz63, dz20)
                    + TRIPLE_PRODUCT(dx43 + dx57, dx64, dx70, dy43 + dy57, dy64, dy70, dz43 + dz57, dz64, dz70)
                    + TRIPLE_PRODUCT(dx14 + dx25, dx61, dx50, dy14 + dy25, dy61, dy50, dz14 + dz25, dz61, dz50);

#undef TRIPLE_PRODUCT

    volume *= twelveth;

    return volume;
}

__host__ __device__ __forceinline__ Real_t CalcElemVolume(Real_t const x[8], Real_t const y[8], Real_t const z[8])
{
    return CalcElemVolume(
        x[0],
        x[1],
        x[2],
        x[3],
        x[4],
        x[5],
        x[6],
        x[7],
        y[0],
        y[1],
        y[2],
        y[3],
        y[4],
        y[5],
        y[6],
        y[7],
        z[0],
        z[1],
        z[2],
        z[3],
        z[4],
        z[5],
        z[6],
        z[7]);
}

void AllocateNodalPersistent(Domain* domain, size_t domNodes)
{
    domain->x.resize(domNodes); /* coordinates */
    domain->y.resize(domNodes);
    domain->z.resize(domNodes);

    domain->xd.resize(domNodes); /* velocities */
    domain->yd.resize(domNodes);
    domain->zd.resize(domNodes);

    domain->xdd.resize(domNodes); /* accelerations */
    domain->ydd.resize(domNodes);
    domain->zdd.resize(domNodes);

    domain->fx.resize(domNodes); /* forces */
    domain->fy.resize(domNodes);
    domain->fz.resize(domNodes);

    domain->nodalMass.resize(domNodes); /* mass */
}

void AllocateElemPersistent(Domain* domain, size_t domElems, size_t padded_domElems)
{
    // domain->matElemlist.resize(domElems) ;  /* material indexset */
    domain->nodelist.resize(8 * padded_domElems); /* elemToNode connectivity */

    domain->lxim.resize(domElems); /* elem connectivity through face */
    domain->lxip.resize(domElems);
    domain->letam.resize(domElems);
    domain->letap.resize(domElems);
    domain->lzetam.resize(domElems);
    domain->lzetap.resize(domElems);

    domain->elemBC.resize(domElems); /* elem face symm/free-surf flag */

    domain->e.resize(domElems); /* energy */
    domain->p.resize(domElems); /* pressure */

    domain->q.resize(domElems); /* q */
    domain->ql.resize(domElems); /* linear term for q */
    domain->qq.resize(domElems); /* quadratic term for q */

    domain->v.resize(domElems); /* relative volume */

    domain->volo.resize(domElems); /* reference volume */
    domain->delv.resize(domElems); /* m_vnew - m_v */
    domain->vdov.resize(domElems); /* volume derivative over volume */

    domain->arealg.resize(domElems); /* elem characteristic length */

    domain->ss.resize(domElems); /* "sound speed" */

    domain->elemMass.resize(domElems); /* mass */
}

void AllocateSymmX(Domain* domain, size_t size)
{
    domain->symmX.resize(size);
}

void AllocateSymmY(Domain* domain, size_t size)
{
    domain->symmY.resize(size);
}

void AllocateSymmZ(Domain* domain, size_t size)
{
    domain->symmZ.resize(size);
}

void terminate_gracefully(void)
{
}

bool InitializeFields(Domain* domain)
{
/* Basic Field Initialization */
#ifdef ALPAKA
    domain->ss.fill(0.);
    domain->e.fill(0.);
    domain->p.fill(0.);
    domain->q.fill(0.);
    domain->v.fill(1.);
    domain->xd.fill(0.);
    domain->yd.fill(0.);
    domain->zd.fill(0.);
    domain->xdd.fill(0.);
    domain->ydd.fill(0.);
    domain->zdd.fill(0.);
    domain->nodalMass.fill(0.);
#else

    thrust::fill(domain->ss.begin(), domain->ss.end(), 0.);
    thrust::fill(domain->e.begin(), domain->e.end(), 0.);
    thrust::fill(domain->p.begin(), domain->p.end(), 0.);
    thrust::fill(domain->q.begin(), domain->q.end(), 0.);
    thrust::fill(domain->v.begin(), domain->v.end(), 1.);

    thrust::fill(domain->xd.begin(), domain->xd.end(), 0.);
    thrust::fill(domain->yd.begin(), domain->yd.end(), 0.);
    thrust::fill(domain->zd.begin(), domain->zd.end(), 0.);

    thrust::fill(domain->xdd.begin(), domain->xdd.end(), 0.);
    thrust::fill(domain->ydd.begin(), domain->ydd.end(), 0.);
    thrust::fill(domain->zdd.begin(), domain->zdd.end(), 0.);

    thrust::fill(domain->nodalMass.begin(), domain->nodalMass.end(), 0.);
#endif

    return true;
}

////////////////////////////////////////////////////////////////////////////////
void Domain::SetupCommBuffers(Int_t edgeNodes)
{
    // allocate a buffer large enough for nodal ghost data
    maxEdgeSize = MAX(this->sizeX, MAX(this->sizeY, this->sizeZ)) + 1;
    maxPlaneSize = CACHE_ALIGN_REAL(maxEdgeSize * maxEdgeSize);
    maxEdgeSize = CACHE_ALIGN_REAL(maxEdgeSize);

    // assume communication to 6 neighbors by default
    m_rowMin = (m_rowLoc == 0) ? 0 : 1;
    m_rowMax = (m_rowLoc == m_tp - 1) ? 0 : 1;
    m_colMin = (m_colLoc == 0) ? 0 : 1;
    m_colMax = (m_colLoc == m_tp - 1) ? 0 : 1;
    m_planeMin = (m_planeLoc == 0) ? 0 : 1;
    m_planeMax = (m_planeLoc == m_tp - 1) ? 0 : 1;

#if USE_MPI
    // account for face communication
    Index_t comBufSize = (m_rowMin + m_rowMax + m_colMin + m_colMax + m_planeMin + m_planeMax) * maxPlaneSize
                         * MAX_FIELDS_PER_MPI_COMM;

    // account for edge communication
    comBufSize
        += ((m_rowMin & m_colMin) + (m_rowMin & m_planeMin) + (m_colMin & m_planeMin) + (m_rowMax & m_colMax)
            + (m_rowMax & m_planeMax) + (m_colMax & m_planeMax) + (m_rowMax & m_colMin) + (m_rowMin & m_planeMax)
            + (m_colMin & m_planeMax) + (m_rowMin & m_colMax) + (m_rowMax & m_planeMin) + (m_colMax & m_planeMin))
           * maxPlaneSize * MAX_FIELDS_PER_MPI_COMM;

    // account for corner communication
    // factor of 16 is so each buffer has its own cache line
    comBufSize += ((m_rowMin & m_colMin & m_planeMin) + (m_rowMin & m_colMin & m_planeMax)
                   + (m_rowMin & m_colMax & m_planeMin) + (m_rowMin & m_colMax & m_planeMax)
                   + (m_rowMax & m_colMin & m_planeMin) + (m_rowMax & m_colMin & m_planeMax)
                   + (m_rowMax & m_colMax & m_planeMin) + (m_rowMax & m_colMax & m_planeMax))
                  * CACHE_COHERENCE_PAD_REAL;

    this->commDataSend = new Real_t[comBufSize];
    this->commDataRecv = new Real_t[comBufSize];

    // pin buffers
    hipHostRegister(this->commDataSend, comBufSize * sizeof(Real_t), 0);
    hipHostRegister(this->commDataRecv, comBufSize * sizeof(Real_t), 0);

    // prevent floating point exceptions
    memset(this->commDataSend, 0, comBufSize * sizeof(Real_t));
    memset(this->commDataRecv, 0, comBufSize * sizeof(Real_t));

    // allocate shadow GPU buffers
    hipMalloc(&this->d_commDataSend, comBufSize * sizeof(Real_t));
    hipMalloc(&this->d_commDataRecv, comBufSize * sizeof(Real_t));

    // prevent floating point exceptions
    hipMemset(this->d_commDataSend, 0, comBufSize * sizeof(Real_t));
    hipMemset(this->d_commDataRecv, 0, comBufSize * sizeof(Real_t));
#endif
}

void SetupConnectivityBC(Domain* domain, int edgeElems)
{
    int domElems = domain->numElem;

    Vector_h<Index_t> lxim_h(domElems);
    Vector_h<Index_t> lxip_h(domElems);
    Vector_h<Index_t> letam_h(domElems);
    Vector_h<Index_t> letap_h(domElems);
    Vector_h<Index_t> lzetam_h(domElems);
    Vector_h<Index_t> lzetap_h(domElems);

    /* set up elemement connectivity information */
    lxim_h[0] = 0;
    for(Index_t i = 1; i < domElems; ++i)
    {
        lxim_h[i] = i - 1;
        lxip_h[i - 1] = i;
    }
    lxip_h[domElems - 1] = domElems - 1;

    for(Index_t i = 0; i < edgeElems; ++i)
    {
        letam_h[i] = i;
        letap_h[domElems - edgeElems + i] = domElems - edgeElems + i;
    }
    for(Index_t i = edgeElems; i < domElems; ++i)
    {
        letam_h[i] = i - edgeElems;
        letap_h[i - edgeElems] = i;
    }

    for(Index_t i = 0; i < edgeElems * edgeElems; ++i)
    {
        lzetam_h[i] = i;
        lzetap_h[domElems - edgeElems * edgeElems + i] = domElems - edgeElems * edgeElems + i;
    }
    for(Index_t i = edgeElems * edgeElems; i < domElems; ++i)
    {
        lzetam_h[i] = i - edgeElems * edgeElems;
        lzetap_h[i - edgeElems * edgeElems] = i;
    }

    /* set up boundary condition information */
    Vector_h<Index_t> elemBC_h(domElems);
    for(Index_t i = 0; i < domElems; ++i)
    {
        elemBC_h[i] = 0; /* clear BCs by default */
    }

    Index_t ghostIdx[6]; // offsets to ghost locations

    for(Index_t i = 0; i < 6; ++i)
    {
        ghostIdx[i] = INT_MIN;
    }

    Int_t pidx = domElems;
    if(domain->m_planeMin != 0)
    {
        ghostIdx[0] = pidx;
        pidx += domain->sizeX * domain->sizeY;
    }

    if(domain->m_planeMax != 0)
    {
        ghostIdx[1] = pidx;
        pidx += domain->sizeX * domain->sizeY;
    }

    if(domain->m_rowMin != 0)
    {
        ghostIdx[2] = pidx;
        pidx += domain->sizeX * domain->sizeZ;
    }

    if(domain->m_rowMax != 0)
    {
        ghostIdx[3] = pidx;
        pidx += domain->sizeX * domain->sizeZ;
    }

    if(domain->m_colMin != 0)
    {
        ghostIdx[4] = pidx;
        pidx += domain->sizeY * domain->sizeZ;
    }

    if(domain->m_colMax != 0)
    {
        ghostIdx[5] = pidx;
    }

    /* symmetry plane or free surface BCs */
    for(Index_t i = 0; i < edgeElems; ++i)
    {
        Index_t planeInc = i * edgeElems * edgeElems;
        Index_t rowInc = i * edgeElems;
        for(Index_t j = 0; j < edgeElems; ++j)
        {
            if(domain->m_planeLoc == 0)
            {
                elemBC_h[rowInc + j] |= ZETA_M_SYMM;
            }
            else
            {
                elemBC_h[rowInc + j] |= ZETA_M_COMM;
                lzetam_h[rowInc + j] = ghostIdx[0] + rowInc + j;
            }

            if(domain->m_planeLoc == domain->m_tp - 1)
            {
                elemBC_h[rowInc + j + domElems - edgeElems * edgeElems] |= ZETA_P_FREE;
            }
            else
            {
                elemBC_h[rowInc + j + domElems - edgeElems * edgeElems] |= ZETA_P_COMM;
                lzetap_h[rowInc + j + domElems - edgeElems * edgeElems] = ghostIdx[1] + rowInc + j;
            }

            if(domain->m_rowLoc == 0)
            {
                elemBC_h[planeInc + j] |= ETA_M_SYMM;
            }
            else
            {
                elemBC_h[planeInc + j] |= ETA_M_COMM;
                letam_h[planeInc + j] = ghostIdx[2] + rowInc + j;
            }

            if(domain->m_rowLoc == domain->m_tp - 1)
            {
                elemBC_h[planeInc + j + edgeElems * edgeElems - edgeElems] |= ETA_P_FREE;
            }
            else
            {
                elemBC_h[planeInc + j + edgeElems * edgeElems - edgeElems] |= ETA_P_COMM;
                letap_h[planeInc + j + edgeElems * edgeElems - edgeElems] = ghostIdx[3] + rowInc + j;
            }

            if(domain->m_colLoc == 0)
            {
                elemBC_h[planeInc + j * edgeElems] |= XI_M_SYMM;
            }
            else
            {
                elemBC_h[planeInc + j * edgeElems] |= XI_M_COMM;
                lxim_h[planeInc + j * edgeElems] = ghostIdx[4] + rowInc + j;
            }

            if(domain->m_colLoc == domain->m_tp - 1)
            {
                elemBC_h[planeInc + j * edgeElems + edgeElems - 1] |= XI_P_FREE;
            }
            else
            {
                elemBC_h[planeInc + j * edgeElems + edgeElems - 1] |= XI_P_COMM;
                lxip_h[planeInc + j * edgeElems + edgeElems - 1] = ghostIdx[5] + rowInc + j;
            }
        }
    }

    domain->elemBC = elemBC_h;
    domain->lxim = lxim_h;
    domain->lxip = lxip_h;
    domain->letam = letam_h;
    domain->letap = letap_h;
    domain->lzetam = lzetam_h;
    domain->lzetap = lzetap_h;
}

void Domain::BuildMesh(
    Int_t nx,
    Int_t edgeNodes,
    Int_t edgeElems,
    Int_t domNodes,
    Int_t padded_domElems,
    Vector_h<Real_t>& x_h,
    Vector_h<Real_t>& y_h,
    Vector_h<Real_t>& z_h,
    Vector_h<Int_t>& nodelist_h)
{
    Index_t meshEdgeElems = m_tp * nx;

    x_h.resize(domNodes);
    y_h.resize(domNodes);
    z_h.resize(domNodes);

    // initialize nodal coordinates
    Index_t nidx = 0;
    Real_t tz = Real_t(1.125) * Real_t(m_planeLoc * nx) / Real_t(meshEdgeElems);
    for(Index_t plane = 0; plane < edgeNodes; ++plane)
    {
        Real_t ty = Real_t(1.125) * Real_t(m_rowLoc * nx) / Real_t(meshEdgeElems);
        for(Index_t row = 0; row < edgeNodes; ++row)
        {
            Real_t tx = Real_t(1.125) * Real_t(m_colLoc * nx) / Real_t(meshEdgeElems);
            for(Index_t col = 0; col < edgeNodes; ++col)
            {
                x_h[nidx] = tx;
                y_h[nidx] = ty;
                z_h[nidx] = tz;
                ++nidx;
                // tx += ds ; // may accumulate roundoff...
                tx = Real_t(1.125) * Real_t(m_colLoc * nx + col + 1) / Real_t(meshEdgeElems);
            }
            // ty += ds ;  // may accumulate roundoff...
            ty = Real_t(1.125) * Real_t(m_rowLoc * nx + row + 1) / Real_t(meshEdgeElems);
        }
        // tz += ds ;  // may accumulate roundoff...
        tz = Real_t(1.125) * Real_t(m_planeLoc * nx + plane + 1) / Real_t(meshEdgeElems);
    }

    x = x_h;
    y = y_h;
    z = z_h;

    nodelist_h.resize(padded_domElems * 8);

    // embed hexehedral elements in nodal point lattice
    Index_t zidx = 0;
    nidx = 0;
    for(Index_t plane = 0; plane < edgeElems; ++plane)
    {
        for(Index_t row = 0; row < edgeElems; ++row)
        {
            for(Index_t col = 0; col < edgeElems; ++col)
            {
                nodelist_h[0 * padded_domElems + zidx] = nidx;
                nodelist_h[1 * padded_domElems + zidx] = nidx + 1;
                nodelist_h[2 * padded_domElems + zidx] = nidx + edgeNodes + 1;
                nodelist_h[3 * padded_domElems + zidx] = nidx + edgeNodes;
                nodelist_h[4 * padded_domElems + zidx] = nidx + edgeNodes * edgeNodes;
                nodelist_h[5 * padded_domElems + zidx] = nidx + edgeNodes * edgeNodes + 1;
                nodelist_h[6 * padded_domElems + zidx] = nidx + edgeNodes * edgeNodes + edgeNodes + 1;
                nodelist_h[7 * padded_domElems + zidx] = nidx + edgeNodes * edgeNodes + edgeNodes;
                ++zidx;
                ++nidx;
            }
            ++nidx;
        }
        nidx += edgeNodes;
    }

    nodelist = nodelist_h; // copies host vector to device vector (throw thrust)
}

Domain* NewDomain(
    char* argv[],
    Int_t numRanks,
    Index_t colLoc,
    Index_t rowLoc,
    Index_t planeLoc,
    Index_t nx,
    int tp,
    bool structured,
    Int_t nr,
    Int_t balance,
    Int_t cost)
{
    Domain* domain = new Domain;
#ifndef ALPAKA
    domain->max_streams = 32;
    domain->streams.resize(domain->max_streams);

    for(Int_t i = 0; i < domain->max_streams; i++)
        hipStreamCreate(&(domain->streams[i]));
#endif
    // TODO get Rid of cuda Event (use some Alpaka function instead)
    // hipEventCreateWithFlags(&domain->time_constraint_computed,hipEventDisableTiming);

    Index_t domElems;
    Index_t domNodes;
    Index_t padded_domElems;
    using std::cout;
    using std::endl;
    Vector_h<Index_t> nodelist_h;
    Vector_h<Real_t> x_h;
    Vector_h<Real_t> y_h;
    Vector_h<Real_t> z_h;
    if(structured)
    {
        domain->m_tp = tp;
        domain->m_numRanks = numRanks;

        domain->m_colLoc = colLoc;
        domain->m_rowLoc = rowLoc;
        domain->m_planeLoc = planeLoc;

        Index_t edgeElems = nx;
        Index_t edgeNodes = edgeElems + 1;
        domain->sizeX = edgeElems;
        domain->sizeY = edgeElems;
        domain->sizeZ = edgeElems;

        domain->numElem = domain->sizeX * domain->sizeY * domain->sizeZ;
        domain->padded_numElem = PAD(domain->numElem, 32);
        domain->numNode = (domain->sizeX + 1) * (domain->sizeY + 1) * (domain->sizeZ + 1);
        domain->padded_numNode = PAD(domain->numNode, 32);
        domElems = domain->numElem;
        domNodes = domain->numNode;
        padded_domElems = domain->padded_numElem;
        AllocateElemPersistent(domain, domElems, padded_domElems);
        AllocateNodalPersistent(domain, domNodes);
        domain->SetupCommBuffers(edgeNodes);

        if(!InitializeFields(domain))
            return NULL;
        domain->BuildMesh(nx, edgeNodes, edgeElems, domNodes, padded_domElems, x_h, y_h, z_h, nodelist_h);
        domain->numSymmX = domain->numSymmY = domain->numSymmZ = 0;

        if(domain->m_colLoc == 0)
            domain->numSymmX = (edgeElems + 1) * (edgeElems + 1);
        if(domain->m_rowLoc == 0)
            domain->numSymmY = (edgeElems + 1) * (edgeElems + 1);
        if(domain->m_planeLoc == 0)
            domain->numSymmZ = (edgeElems + 1) * (edgeElems + 1);
        AllocateSymmX(domain, edgeNodes * edgeNodes);
        AllocateSymmY(domain, edgeNodes * edgeNodes);
        AllocateSymmZ(domain, edgeNodes * edgeNodes);

        /* set up symmetry nodesets */

        Vector_h<Index_t> symmX_h(domain->symmX.size());
        Vector_h<Index_t> symmY_h(domain->symmY.size());
        Vector_h<Index_t> symmZ_h(domain->symmZ.size());

        Int_t nidx = 0;
        for(Index_t i = 0; i < edgeNodes; ++i)
        {
            Index_t planeInc = i * edgeNodes * edgeNodes;
            Index_t rowInc = i * edgeNodes;
            for(Index_t j = 0; j < edgeNodes; ++j)
            {
                if(domain->m_planeLoc == 0)
                {
                    symmZ_h[nidx] = rowInc + j;
                }
                if(domain->m_rowLoc == 0)
                {
                    symmY_h[nidx] = planeInc + j;
                }
                if(domain->m_colLoc == 0)
                {
                    symmX_h[nidx] = planeInc + j * edgeNodes;
                }
                ++nidx;
            }
        }

        if(domain->m_planeLoc == 0)
            domain->symmZ = symmZ_h;
        if(domain->m_rowLoc == 0)
            domain->symmY = symmY_h;
        if(domain->m_colLoc == 0)
            domain->symmX = symmX_h;

        SetupConnectivityBC(domain, edgeElems);
    }
    else
    {
        FILE* fp;
        int ee, en;

        if((fp = fopen(argv[2], "r")) == 0)
        {
            printf("could not open file %s\n", argv[2]);
            exit(LFileError);
        }

        bool fsuccess;
        fsuccess = fscanf(fp, "%d %d", &ee, &en);
        domain->numElem = Index_t(ee);
        domain->padded_numElem = PAD(domain->numElem, 32);

        domain->numNode = Index_t(en);
        domain->padded_numNode = PAD(domain->numNode, 32);

        domElems = domain->numElem;
        domNodes = domain->numNode;
        padded_domElems = domain->padded_numElem;

        AllocateElemPersistent(domain, domElems, padded_domElems);
        AllocateNodalPersistent(domain, domNodes);

        InitializeFields(domain);

        /* initialize nodal coordinates */
        x_h.resize(domNodes);
        y_h.resize(domNodes);
        z_h.resize(domNodes);

        for(Index_t i = 0; i < domNodes; ++i)
        {
            double px, py, pz;
            fsuccess = fscanf(fp, "%lf %lf %lf", &px, &py, &pz);
            x_h[i] = Real_t(px);
            y_h[i] = Real_t(py);
            z_h[i] = Real_t(pz);
        }
        domain->x = x_h;
        domain->y = y_h;
        domain->z = z_h;

        /* embed hexehedral elements in nodal point lattice */
        nodelist_h.resize(padded_domElems * 8);
        for(Index_t zidx = 0; zidx < domElems; ++zidx)
        {
            for(Index_t ni = 0; ni < Index_t(8); ++ni)
            {
                int n;
                fsuccess = fscanf(fp, "%d", &n);
                nodelist_h[ni * padded_domElems + zidx] = Index_t(n);
            }
        }
        domain->nodelist = nodelist_h;

        /* set up face-based element neighbors */
        Vector_h<Index_t> lxim_h(domElems);
        Vector_h<Index_t> lxip_h(domElems);
        Vector_h<Index_t> letam_h(domElems);
        Vector_h<Index_t> letap_h(domElems);
        Vector_h<Index_t> lzetam_h(domElems);
        Vector_h<Index_t> lzetap_h(domElems);

        for(Index_t i = 0; i < domElems; ++i)
        {
            int xi_m, xi_p, eta_m, eta_p, zeta_m, zeta_p;
            fsuccess = fscanf(fp, "%d %d %d %d %d %d", &xi_m, &xi_p, &eta_m, &eta_p, &zeta_m, &zeta_p);

            lxim_h[i] = Index_t(xi_m);
            lxip_h[i] = Index_t(xi_p);
            letam_h[i] = Index_t(eta_m);
            letap_h[i] = Index_t(eta_p);
            lzetam_h[i] = Index_t(zeta_m);
            lzetap_h[i] = Index_t(zeta_p);
        }

        domain->lxim = lxim_h;
        domain->lxip = lxip_h;
        domain->letam = letam_h;
        domain->letap = letap_h;
        domain->lzetam = lzetam_h;
        domain->lzetap = lzetap_h;

        /* set up X symmetry nodeset */

        fsuccess = fscanf(fp, "%d", &domain->numSymmX);
        Vector_h<Index_t> symmX_h(domain->numSymmX);
        for(Index_t i = 0; i < domain->numSymmX; ++i)
        {
            int n;
            fsuccess = fscanf(fp, "%d", &n);
            symmX_h[i] = Index_t(n);
        }
        domain->symmX = symmX_h;

        fsuccess = fscanf(fp, "%d", &domain->numSymmY);
        Vector_h<Index_t> symmY_h(domain->numSymmY);
        for(Index_t i = 0; i < domain->numSymmY; ++i)
        {
            int n;
            fsuccess = fscanf(fp, "%d", &n);
            symmY_h[i] = Index_t(n);
        }
        domain->symmY = symmY_h;

        fsuccess = fscanf(fp, "%d", &domain->numSymmZ);
        Vector_h<Index_t> symmZ_h(domain->numSymmZ);
        for(Index_t i = 0; i < domain->numSymmZ; ++i)
        {
            int n;
            fsuccess = fscanf(fp, "%d", &n);
            symmZ_h[i] = Index_t(n);
        }
        domain->symmZ = symmZ_h;

        /* set up free surface nodeset */
        Index_t numFreeSurf;
        fsuccess = fscanf(fp, "%d", &numFreeSurf);
        Vector_h<Index_t> freeSurf_h(numFreeSurf);
        for(Index_t i = 0; i < numFreeSurf; ++i)
        {
            int n;
            fsuccess = fscanf(fp, "%d", &n);
            freeSurf_h[i] = Index_t(n);
        }
        printf("%c\n", fsuccess); // nothing
        fclose(fp);

        /* set up boundary condition information */
        Vector_h<Index_t> elemBC_h(domElems);
        Vector_h<Index_t> surfaceNode_h(domNodes);

        for(Index_t i = 0; i < domain->numElem; ++i)
        {
            elemBC_h[i] = 0;
        }

        for(Index_t i = 0; i < domain->numNode; ++i)
        {
            surfaceNode_h[i] = 0;
        }

        for(Index_t i = 0; i < domain->numSymmX; ++i)
        {
            surfaceNode_h[symmX_h[i]] = 1;
        }

        for(Index_t i = 0; i < domain->numSymmY; ++i)
        {
            surfaceNode_h[symmY_h[i]] = 1;
        }

        for(Index_t i = 0; i < domain->numSymmZ; ++i)
        {
            surfaceNode_h[symmZ_h[i]] = 1;
        }

        for(Index_t zidx = 0; zidx < domain->numElem; ++zidx)
        {
            Int_t mask = 0;

            for(Index_t ni = 0; ni < 8; ++ni)
            {
                mask |= (surfaceNode_h[nodelist_h[ni * domain->padded_numElem + zidx]] << ni);
            }

            if((mask & 0x0f) == 0x0f)
                elemBC_h[zidx] |= ZETA_M_SYMM;
            if((mask & 0xf0) == 0xf0)
                elemBC_h[zidx] |= ZETA_P_SYMM;
            if((mask & 0x33) == 0x33)
                elemBC_h[zidx] |= ETA_M_SYMM;
            if((mask & 0xcc) == 0xcc)
                elemBC_h[zidx] |= ETA_P_SYMM;
            if((mask & 0x99) == 0x99)
                elemBC_h[zidx] |= XI_M_SYMM;
            if((mask & 0x66) == 0x66)
                elemBC_h[zidx] |= XI_P_SYMM;
        }

        for(Index_t zidx = 0; zidx < domain->numElem; ++zidx)
        {
            if(elemBC_h[zidx] == (XI_M_SYMM | ETA_M_SYMM | ZETA_M_SYMM))
            {
                domain->octantCorner = zidx;
                break;
            }
        }

        for(Index_t i = 0; i < domain->numNode; ++i)
        {
            surfaceNode_h[i] = 0;
        }

        for(Index_t i = 0; i < numFreeSurf; ++i)
        {
            surfaceNode_h[freeSurf_h[i]] = 1;
        }

        for(Index_t zidx = 0; zidx < domain->numElem; ++zidx)
        {
            Int_t mask = 0;

            for(Index_t ni = 0; ni < 8; ++ni)
            {
                mask |= (surfaceNode_h[nodelist_h[ni * domain->padded_numElem + zidx]] << ni);
            }

            if((mask & 0x0f) == 0x0f)
                elemBC_h[zidx] |= ZETA_M_SYMM;
            if((mask & 0xf0) == 0xf0)
                elemBC_h[zidx] |= ZETA_P_SYMM;
            if((mask & 0x33) == 0x33)
                elemBC_h[zidx] |= ETA_M_SYMM;
            if((mask & 0xcc) == 0xcc)
                elemBC_h[zidx] |= ETA_P_SYMM;
            if((mask & 0x99) == 0x99)
                elemBC_h[zidx] |= XI_M_SYMM;
            if((mask & 0x66) == 0x66)
                elemBC_h[zidx] |= XI_P_SYMM;
        }

        domain->elemBC = elemBC_h;

        /* deposit energy */
        Real_t arg[] = {3.948746e+7};
        domain->e.changeValue(domain->octantCorner, 1, &arg[0]);
    }
    /* set up node-centered indexing of elements */
    Vector_h<Index_t> nodeElemCount_h(domNodes);

    for(Index_t i = 0; i < domNodes; ++i)
    {
        nodeElemCount_h[i] = 0;
    }

    for(Index_t i = 0; i < domElems; ++i)
    {
        for(Index_t j = 0; j < 8; ++j)
        {
            ++(nodeElemCount_h[nodelist_h[j * padded_domElems + i]]);
        }
    }

    Vector_h<Index_t> nodeElemStart_h(domNodes);

    nodeElemStart_h[0] = 0;
    for(Index_t i = 1; i < domNodes; ++i)
    {
        nodeElemStart_h[i] = nodeElemStart_h[i - 1] + nodeElemCount_h[i - 1];
    }

    Vector_h<Index_t> nodeElemCornerList_h(nodeElemStart_h[domNodes - 1] + nodeElemCount_h[domNodes - 1]);

    for(Index_t i = 0; i < domNodes; ++i)
    {
        nodeElemCount_h[i] = 0;
    }

    for(Index_t j = 0; j < 8; ++j)
    {
        for(Index_t i = 0; i < domElems; ++i)
        {
            Index_t m = nodelist_h[padded_domElems * j + i];
            Index_t k = padded_domElems * j + i;
            Index_t offset = nodeElemStart_h[m] + nodeElemCount_h[m];
            nodeElemCornerList_h[offset] = k;
            ++(nodeElemCount_h[m]);
        }
    }

    Index_t clSize = nodeElemStart_h[domNodes - 1] + nodeElemCount_h[domNodes - 1];
    for(Index_t i = 0; i < clSize; ++i)
    {
        Index_t clv = nodeElemCornerList_h[i];
        if((clv < 0) || (clv > padded_domElems * 8))
        {
            fprintf(
                stderr,
                "AllocateNodeElemIndexes(): nodeElemCornerList entry out "
                "of range!\n");
            exit(1);
        }
    }

    domain->nodeElemStart = nodeElemStart_h;
    domain->nodeElemCount = nodeElemCount_h;
    domain->nodeElemCornerList = nodeElemCornerList_h;

    /* Create a material IndexSet (entire domain same material for now) */
    Vector_h<Index_t> matElemlist_h(domElems);
    for(Index_t i = 0; i < domElems; ++i)
    {
        matElemlist_h[i] = i;
    }
    domain->matElemlist = matElemlist_h;
    Vector_h<Real_t> constraints_h(4, 1e20);
    constraints_h[2] = -1.0;
    constraints_h[3] = -1.0;
    domain->constraints_h = constraints_h;
    domain->constraints_d = constraints_h;
    /*domain->dtcourant_d = dtcourant;
    domain->dthydro_d = dthydro;
    domain->bad_vol_d = bad_vol;
    domain->bad_q_d = bad_q;*/
    /* hipHostMalloc(&domain->dtcourant_h,sizeof(Real_t),0);
    hipHostMalloc(&domain->dthydro_h,sizeof(Real_t),0);
    hipHostMalloc(&domain->bad_vol_h,sizeof(Index_t),0);//check
    hipHostMalloc(&domain->bad_q_h,sizeof(Index_t),0);//check*/
    /* initialize material parameters */
    domain->time_h = Real_t(0.);
    domain->dtfixed = Real_t(-1.0e-6);
    domain->deltatimemultlb = Real_t(1.1);
    domain->deltatimemultub = Real_t(1.2);
    domain->stoptime = Real_t(1.0e-2);
    domain->dtmax = Real_t(1.0e-2);
    domain->cycle = 0;

    domain->e_cut = Real_t(1.0e-7);
    domain->p_cut = Real_t(1.0e-7);
    domain->q_cut = Real_t(1.0e-7);
    domain->u_cut = Real_t(1.0e-7);
    domain->v_cut = Real_t(1.0e-10);

    domain->hgcoef = Real_t(3.0);
    domain->ss4o3 = Real_t(4.0) / Real_t(3.0);

    domain->qstop = Real_t(1.0e+12);
    domain->monoq_max_slope = Real_t(1.0);
    domain->monoq_limiter_mult = Real_t(2.0);
    domain->qlc_monoq = Real_t(0.5);
    domain->qqc_monoq = Real_t(2.0) / Real_t(3.0);
    domain->qqc = Real_t(2.0);

    domain->pmin = Real_t(0.);
    domain->emin = Real_t(-1.0e+15);

    domain->dvovmax = Real_t(0.1);

    domain->eosvmax = Real_t(1.0e+9);
    domain->eosvmin = Real_t(1.0e-9);

    domain->refdens = Real_t(1.0);
    /* initialize field data */
    Vector_h<Real_t> nodalMass_h(domNodes);
    Vector_h<Real_t> volo_h(domElems);
    Vector_h<Real_t> elemMass_h(domElems);
    for(Index_t i = 0; i < domElems; ++i)
    {
        Real_t x_local[8], y_local[8], z_local[8];
        for(Index_t lnode = 0; lnode < 8; ++lnode)
        {
            Index_t gnode = nodelist_h[lnode * padded_domElems + i];
            x_local[lnode] = x_h[gnode];
            y_local[lnode] = y_h[gnode];
            z_local[lnode] = z_h[gnode];
        }

        // volume calculations
        Real_t volume = CalcElemVolume(x_local, y_local, z_local);
        volo_h[i] = volume;
        elemMass_h[i] = volume;
        for(Index_t j = 0; j < 8; ++j)
        {
            Index_t gnode = nodelist_h[j * padded_domElems + i];
            nodalMass_h[gnode] += volume / Real_t(8.0);
        }
    }
    domain->nodalMass = nodalMass_h;
    domain->volo = volo_h;
    domain->elemMass = elemMass_h;

    /* deposit energy */
    domain->octantCorner = 0;
    // deposit initial energy
    // An energy of 3.948746e+7 is correct for a problem with
    // 45 zones along a side - we need to scale it
    Real_t const ebase = 3.948746e+7;
    Real_t scale = (nx * domain->m_tp) / 45.0;
    Real_t einit = ebase * scale * scale * scale;
    // Real_t einit = ebase;
    if(domain->m_rowLoc + domain->m_colLoc + domain->m_planeLoc == 0)
    {
        // Dump into the first zone (which we know is in the corner)
        // of the domain that sits at the origin
#ifdef ALPAKA
        Real_t arg[] = {einit};
        domain->e.changeValue(0, 1, &arg[0]);
#else
        domain->e[0] = einit;
#endif
    }
    // set initial deltatime base on analytic CFL calculation
    domain->deltatime_h = (.5 * cbrt(domain->volo.accessIndex(1))) / sqrt(2 * einit);
    domain->cost = cost;
    domain->regNumList.resize(domain->numElem); // material indexset
    domain->regElemlist.resize(domain->numElem); // material indexset
    domain->regCSR.resize(nr);
    domain->regReps.resize(nr);
    domain->regSorted.resize(nr);
    // Setup region index sets. For now, these are constant sized
    // throughout the run, but could be changed every cycle to
    // simulate effects of ALE on the lagrange solver

    domain->CreateRegionIndexSets(nr, balance);

    /*cout << "[DEBUG] Printing domain variables:\n\ndomain->m_tp: " <<
    domain->m_tp << endl << "domain->m_numRanks: " << domain->m_numRanks << endl;
    cout << "domain->m_colLoc: " << domain->m_colLoc << endl << "domain->m_rowLoc:
    " << domain->m_rowLoc << endl << "domain->m_planeLoc: " << domain->m_planeLoc
    << endl; cout << "domain->numElem: " << domain->numElem << endl <<
    "domain->padded_numElem: " << domain->padded_numElem << endl <<
    "domain->numNode: " << domain->numNode << endl; cout <<
    "domain->padded_numNode: " << domain->padded_numNode << endl <<
    "domain->numSymmX: " << domain->numSymmX << endl << "domain->numSymmY: " <<
    domain->numSymmY << endl; cout << "domain->numSymmZ: " << domain->numSymmZ <<
    endl << "domain->symmZ: " << domain->symmZ[0] << endl << "domain->symmX: " <<
    domain->symmX[0] << endl << "domain->symmY: " << domain->symmY[0] << endl;
    cout << "domain->nodalMass[100]: " << domain->nodalMass[100] << endl <<
    "domain->volo[0]: " << domain->volo[100] << endl << "domain->elemMass[0]: " <<
    domain->elemMass[100] << endl; cout << "domain->dtcourant_d: " <<
    domain->dtcourant_d[0] << endl; cout << "domain->deltatime_h: " <<
    domain->deltatime_h << endl << "domain->cost: " << domain->cost << endl;
    cout << "nr: " << nr << endl << "balance: " << balance << endl;*/
    // exit(1);
    return domain;
}

/*******************	to support region	*********************/
void Domain::sortRegions(Vector_h<Int_t>& regReps_h, Vector_h<Index_t>& regSorted_h)
{
    Index_t temp;
    Vector_h<Index_t> regIndex;
    regIndex.resize(numReg);
    for(int i = 0; i < numReg; i++)
        regIndex[i] = i;

    for(int i = 0; i < numReg - 1; i++)
        for(int j = 0; j < numReg - i - 1; j++)
            if(regReps_h[j] < regReps_h[j + 1])
            {
                temp = regReps_h[j];
                regReps_h[j] = regReps_h[j + 1];
                regReps_h[j + 1] = temp;

                temp = regElemSize[j];
                regElemSize[j] = regElemSize[j + 1];
                regElemSize[j + 1] = temp;

                temp = regIndex[j];
                regIndex[j] = regIndex[j + 1];
                regIndex[j + 1] = temp;
            }
    for(int i = 0; i < numReg; i++)
        regSorted_h[regIndex[i]] = i;
}

// simple function for int pow x^y, y >= 0
Int_t POW(Int_t x, Int_t y)
{
    Int_t res = 1;
    for(Int_t i = 0; i < y; i++)
        res *= x;
    return res;
}

void Domain::CreateRegionIndexSets(Int_t nr, Int_t b)
{
#if USE_MPI
    Index_t myRank;
    MPI_Comm_rank(MPI_COMM_WORLD, &myRank);
    srand(myRank);
#else
    srand(0);
    Index_t myRank = 0;
#endif
    numReg = nr;
    balance = b;

    regElemSize = new Int_t[numReg];
    Index_t nextIndex = 0;

    Vector_h<Int_t> regCSR_h(regCSR.size()); // records the begining and end of each region
    Vector_h<Int_t> regReps_h(regReps.size()); // records the rep number per region
    Vector_h<Index_t> regNumList_h(regNumList.size()); // Region number per domain element
    Vector_h<Index_t> regElemlist_h(regElemlist.size()); // region indexset
    Vector_h<Index_t> regSorted_h(regSorted.size()); // keeps index of sorted regions

    // if we only have one region just fill it
    //  Fill out the regNumList with material numbers, which are always
    //  the region index plus one
    if(numReg == 1)
    {
        while(nextIndex < numElem)
        {
            regNumList_h[nextIndex] = 1;
            nextIndex++;
        }
        regElemSize[0] = 0;
    }
    // If we have more than one region distribute the elements.
    else
    {
        Int_t regionNum;
        Int_t regionVar;
        Int_t lastReg = -1;
        Int_t binSize;
        Int_t elements;
        Index_t runto = 0;
        Int_t costDenominator = 0;
        Int_t* regBinEnd = new Int_t[numReg];
        // Determine the relative weights of all the regions.
        for(Index_t i = 0; i < numReg; ++i)
        {
            regElemSize[i] = 0;
            costDenominator += POW((i + 1), balance); // Total cost of all regions
            regBinEnd[i] = costDenominator; // Chance of hitting a given region is (regBinEnd[i]
                                            // - regBinEdn[i-1])/costDenominator
        }
        // Until all elements are assigned
        while(nextIndex < numElem)
        {
            // pick the region
            regionVar = rand() % costDenominator;
            Index_t i = 0;
            while(regionVar >= regBinEnd[i])
                i++;
            // rotate the regions based on MPI rank.  Rotation is Rank % NumRegions
            regionNum = ((i + myRank) % numReg) + 1;
            // make sure we don't pick the same region twice in a row
            while(regionNum == lastReg)
            {
                regionVar = rand() % costDenominator;
                i = 0;
                while(regionVar >= regBinEnd[i])
                    i++;
                regionNum = ((i + myRank) % numReg) + 1;
            }
            // Pick the bin size of the region and determine the number of elements.
            binSize = rand() % 1000;
            if(binSize < 773)
            {
                elements = rand() % 15 + 1;
            }
            else if(binSize < 937)
            {
                elements = rand() % 16 + 16;
            }
            else if(binSize < 970)
            {
                elements = rand() % 32 + 32;
            }
            else if(binSize < 974)
            {
                elements = rand() % 64 + 64;
            }
            else if(binSize < 978)
            {
                elements = rand() % 128 + 128;
            }
            else if(binSize < 981)
            {
                elements = rand() % 256 + 256;
            }
            else
                elements = rand() % 1537 + 512;
            runto = elements + nextIndex;
            // Store the elements.  If we hit the end before we run out of elements
            // then just stop.
            while(nextIndex < runto && nextIndex < numElem)
            {
                regNumList_h[nextIndex] = regionNum;
                nextIndex++;
            }
            lastReg = regionNum;
        }
    }
    // Convert regNumList to region index sets
    // First, count size of each region
    for(Index_t i = 0; i < numElem; ++i)
    {
        int r = regNumList_h[i] - 1; // region index == regnum-1
        regElemSize[r]++;
    }

    Index_t rep;
    // Second, allocate each region index set
    for(Index_t r = 0; r < numReg; ++r)
    {
        if(r < numReg / 2)
            rep = 1;
        else if(r < (numReg - (numReg + 15) / 20))
            rep = 1 + cost;
        else
            rep = 10 * (1 + cost);
        regReps_h[r] = rep;
    }

    sortRegions(regReps_h, regSorted_h);

    regCSR_h[0] = 0;
    // Second, allocate each region index set
    for(Index_t i = 1; i < numReg; ++i)
    {
        regCSR_h[i] = regCSR_h[i - 1] + regElemSize[i - 1];
    }

    // Third, fill index sets
    for(Index_t i = 0; i < numElem; ++i)
    {
        Index_t r = regSorted_h[regNumList_h[i] - 1]; // region index == regnum-1
        regElemlist_h[regCSR_h[r]] = i;
        regCSR_h[r]++;
    }

    // Copy to device
    regCSR = regCSR_h; // records the begining and end of each region
    regReps = regReps_h; // records the rep number per region
    regNumList = regNumList_h; // Region number per domain element
    regElemlist = regElemlist_h; // region indexset
    regSorted = regSorted_h; // keeps index of sorted regions

} // end of create function

inline void TimeIncrement(Domain* domain)
{
    // To make sure dtcourant and dthydro have been updated on host
    Real_t targetdt = domain->stoptime - domain->time_h;
    domain->constraints_h = domain->constraints_d; // copy all constraint values from device
    if((domain->dtfixed <= Real_t(0.0)) && (domain->cycle != Int_t(0)))
    {
        Real_t ratio;

        /* This will require a reduction in parallel */
        Real_t gnewdt = Real_t(1.0e+20);
        Real_t newdt;
        Real_t dtcourant_d_val = domain->constraints_h[0];
        Real_t dthydro_d_val = domain->constraints_h[1];
        if(dtcourant_d_val < gnewdt)
        {
            gnewdt = dtcourant_d_val / Real_t(2.0);
        }
        if(dthydro_d_val < gnewdt)
        {
            gnewdt = dthydro_d_val * Real_t(2.0) / Real_t(3.0);
        }

#if USE_MPI
        MPI_Allreduce(&gnewdt, &newdt, 1, ((sizeof(Real_t) == 4) ? MPI_FLOAT : MPI_DOUBLE), MPI_MIN, MPI_COMM_WORLD);
#else
        newdt = gnewdt;
#endif

        Real_t olddt = domain->deltatime_h;
        ratio = newdt / olddt;
        if(ratio >= Real_t(1.0))
        {
            if(ratio < domain->deltatimemultlb)
            {
                newdt = olddt;
            }
            else if(ratio > domain->deltatimemultub)
            {
                newdt = olddt * domain->deltatimemultub;
            }
        }

        if(newdt > domain->dtmax)
        {
            newdt = domain->dtmax;
        }
        domain->deltatime_h = newdt;
    }

    /* TRY TO PREVENT VERY SMALL SCALING ON THE NEXT CYCLE */
    if((targetdt > domain->deltatime_h) && (targetdt < (Real_t(4.0) * domain->deltatime_h / Real_t(3.0))))
    {
        targetdt = Real_t(2.0) * domain->deltatime_h / Real_t(3.0);
    }

    if(targetdt < domain->deltatime_h)
    {
        domain->deltatime_h = targetdt;
    }

    domain->time_h += domain->deltatime_h;

    ++domain->cycle;
}

inline void CalcVolumeForceForElems(Real_t const hgcoef, Domain* domain)
{
    Index_t numElem = domain->numElem;
    Index_t padded_numElem = domain->padded_numElem;

#ifdef DOUBLE_PRECISION
#    ifdef ALPAKA
    Vector_d<Real_t> fx_elem(padded_numElem * 8);

    Vector_d<Real_t> fy_elem(padded_numElem * 8);
    Vector_d<Real_t> fz_elem(padded_numElem * 8);
#    else
    Vector_d<Real_t>* fx_elem = Allocator<Vector_d<Real_t>>::allocate(padded_numElem * 8);
    Vector_d<Real_t>* fy_elem = Allocator<Vector_d<Real_t>>::allocate(padded_numElem * 8);
    Vector_d<Real_t>* fz_elem = Allocator<Vector_d<Real_t>>::allocate(padded_numElem * 8);
#    endif
#else
    thrust::fill(domain->fx.begin(), domain->fx.end(), 0.);
    thrust::fill(domain->fy.begin(), domain->fy.end(), 0.);
    thrust::fill(domain->fz.begin(), domain->fz.end(), 0.);
#endif

    int num_threads = numElem;
    int const block_size = 64;
    int dimGrid = PAD_DIV(num_threads, block_size);

    bool const hourg_gt_zero = hgcoef > Real_t(0.0);
#ifdef ALPAKA

    using CalcElemForce = lulesh_port_kernels::CalcVolumeForceForElems_kernel_class;
    CalcElemForce ElemForceKernel(
        domain->volo.raw(),
        domain->v.raw(),
        domain->p.raw(),
        domain->q.raw(),
        hgcoef,
        numElem,
        padded_numElem,
        domain->nodelist.raw(),
        domain->ss.raw(),
        domain->elemMass.raw(),
        domain->x.raw(),
        domain->y.raw(),
        domain->z.raw(),
        domain->xd.raw(),
        domain->yd.raw(),
        domain->zd.raw(),
        fx_elem.raw(),
        fy_elem.raw(),
        fz_elem.raw(),
        domain->constraints_d.raw(),
        num_threads,
        hourg_gt_zero);

    using Dim2 = alpaka::DimInt<2>;
    using Idx = std::size_t;
    using Vec2 = alpaka::Vec<Dim2, Idx>;

    alpaka_utils::alpakaExecuteBaseKernel<Dim2, Idx>(ElemForceKernel, Vec2{block_size, dimGrid}, true);
    // hipDeviceSynchronize();
    // Vector_h h_n(domain->ss);

#else
    if(hourg_gt_zero)
    {
        CalcVolumeForceForElems_kernel<true><<<dimGrid, block_size>>>(
            domain->volo.raw(),
            domain->v.raw(),
            domain->p.raw(),
            domain->q.raw(),
            hgcoef,
            numElem,
            padded_numElem,
            domain->nodelist.raw(),
            domain->ss.raw(),
            domain->elemMass.raw(),
            domain->x.raw(),
            domain->y.raw(),
            domain->z.raw(),
            domain->xd.raw(),
            domain->yd.raw(),
            domain->zd.raw(),
#    ifdef DOUBLE_PRECISION
            fx_elem->raw(),
            fy_elem->raw(),
            fz_elem->raw(),
#    else
            domain->fx.raw(),
            domain->fy.raw(),
            domain->fz.raw(),
#    endif
            domain->bad_vol_h,
            num_threads);
    }
    else
    {
        CalcVolumeForceForElems_kernel<false><<<dimGrid, block_size>>>(
            domain->volo.raw(),
            domain->v.raw(),
            domain->p.raw(),
            domain->q.raw(),
            hgcoef,
            numElem,
            padded_numElem,
            domain->nodelist.raw(),
            domain->ss.raw(),
            domain->elemMass.raw(),
            domain->x.raw(),
            domain->y.raw(),
            domain->z.raw(),
            domain->xd.raw(),
            domain->yd.raw(),
            domain->zd.raw(),

#    ifdef DOUBLE_PRECISION
            fx_elem->raw(),
            fy_elem->raw(),
            fz_elem->raw(),
#    else
            domain->fx.raw(),
            domain->fy.raw(),
            domain->fz.raw(),
#    endif
            domain->bad_vol_h,
            num_threads);
    }
#endif // endif ALPAKA

#ifdef DOUBLE_PRECISION
    num_threads = domain->numNode;

    // Launch boundary nodes first
    dimGrid = PAD_DIV(num_threads, block_size);
#    ifdef ALPAKA
    using AddNodeForce = lulesh_port_kernels::AddNodeForcesFromElems_kernel_class;
    AddNodeForce NodeForceKernel(
        domain->numNode,
        domain->padded_numNode,
        domain->nodeElemCount.raw(),
        domain->nodeElemStart.raw(),
        domain->nodeElemCornerList.raw(),
        fx_elem.raw(),
        fy_elem.raw(),
        fz_elem.raw(),
        domain->fx.raw(),
        domain->fy.raw(),
        domain->fz.raw(),
        num_threads);
    // cudaCheckError();
    using Dim2 = alpaka::DimInt<2>;
    using Idx = std::size_t;
    using Vec2 = alpaka::Vec<Dim2, Idx>;

    alpaka_utils::alpakaExecuteBaseKernel<Dim2, Idx>(NodeForceKernel, Vec2{block_size, dimGrid}, true);
    // cudaCheckError();
    // hipDeviceSynchronize();

#    else

    AddNodeForcesFromElems_kernel<<<dimGrid, block_size>>>(
        domain->numNode,
        domain->padded_numNode,
        domain->nodeElemCount.raw(),
        domain->nodeElemStart.raw(),
        domain->nodeElemCornerList.raw(),
        fx_elem.raw(),
        fy_elem.raw(),
        fz_elem.raw(),
        domain->fx.raw(),
        domain->fy.raw(),
        domain->fz.raw(),
        num_threads);
    cudaCheckError();
    hipDeviceSynchronize();
    Allocator<Vector_d<Real_t>>::free(fx_elem, padded_numElem * 8);
    Allocator<Vector_d<Real_t>>::free(fy_elem, padded_numElem * 8);
    Allocator<Vector_d<Real_t>>::free(fz_elem, padded_numElem * 8);
#    endif

#endif // ifdef DOUBLE_PRECISION
    return;
};

inline void CalcVolumeForceForElems(Domain* domain)
{
    Real_t const hgcoef = domain->hgcoef;

    CalcVolumeForceForElems(hgcoef, domain);
    // cudaCheckError();

    // CalcVolumeForceForElems_warp_per_4cell(hgcoef,domain);
};

inline void checkErrors(Domain* domain, int its, int myRank)
{
    auto bad_vol = domain->constraints_h[2];
    auto bad_q = domain->constraints_h[3];
    if(bad_vol != -1.0)
    {
        printf("Rank %i: Volume Error in cell %d at iteration %d\n", myRank, bad_vol, its);
        exit(VolumeError);
    }

    if(bad_q != -1.0)
    {
        printf("Rank %i: Q Error in cell %d at iteration %d\n", myRank, bad_q, its);
        exit(QStopError);
    }
}

inline void CalcForceForNodes(Domain* domain)
{
#if USE_MPI
    CommRecv(*domain, MSG_COMM_SBN, 3, domain->sizeX + 1, domain->sizeY + 1, domain->sizeZ + 1, true, false);
#endif

    CalcVolumeForceForElems(domain);
    // cudaCheckError();
    //  moved here from the main loop to allow async execution with GPU work
    TimeIncrement(domain);

#if USE_MPI
    // initialize pointers
    domain->d_fx = domain->fx.raw();
    domain->d_fy = domain->fy.raw();
    domain->d_fz = domain->fz.raw();

    Domain_member fieldData[3];
    fieldData[0] = &Domain::get_fx;
    fieldData[1] = &Domain::get_fy;
    fieldData[2] = &Domain::get_fz;

    CommSendGpu(
        *domain,
        MSG_COMM_SBN,
        3,
        fieldData,
        domain->sizeX + 1,
        domain->sizeY + 1,
        domain->sizeZ + 1,
        true,
        false,
        domain->streams[2]);
    CommSBNGpu(*domain, 3, fieldData, &domain->streams[2]);
#endif
}

__global__ void CalcAccelerationForNodes_kernel(
    int numNode,
    Real_t* xdd,
    Real_t* ydd,
    Real_t* zdd,
    Real_t* fx,
    Real_t* fy,
    Real_t* fz,
    Real_t* nodalMass)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < numNode)
    {
        Real_t one_over_nMass = Real_t(1.) / nodalMass[tid];
        xdd[tid] = fx[tid] * one_over_nMass;
        ydd[tid] = fy[tid] * one_over_nMass;
        zdd[tid] = fz[tid] * one_over_nMass;
    }
}

inline void CalcAccelerationForNodes(Domain* domain)
{
    int const dimBlock = 128;
    int dimGrid = PAD_DIV(static_cast<int>(domain->numNode), dimBlock);
    // cudaCheckError();

#ifdef ALPAKA
    using CalcAccelerationNodes = lulesh_port_kernels::CalcAccelerationForNodes_kernel_class;
    CalcAccelerationNodes CalcAccNodeKernel(
        domain->numNode,
        domain->xdd.raw(),
        domain->ydd.raw(),
        domain->zdd.raw(),
        domain->fx.raw(),
        domain->fy.raw(),
        domain->fz.raw(),
        domain->nodalMass.raw());
    // cudaCheckError();
    using Dim2 = alpaka::DimInt<2>;
    using Idx = std::size_t;
    using Vec2 = alpaka::Vec<Dim2, Idx>;
    alpaka_utils::alpakaExecuteBaseKernel<Dim2, Idx>(CalcAccNodeKernel, Vec2{dimBlock, dimGrid}, true);
    // cudaCheckError();
    // hipDeviceSynchronize();
#else
    CalcAccelerationForNodes_kernel<<<dimGrid, dimBlock>>>(
        domain->numNode,
        domain->xdd.raw(),
        domain->ydd.raw(),
        domain->zdd.raw(),
        domain->fx.raw(),
        domain->fy.raw(),
        domain->fz.raw(),
        domain->nodalMass.raw());
#endif
}

__global__ void ApplyAccelerationBoundaryConditionsForNodes_kernel(int numNodeBC, Real_t* xyzdd, Index_t* symm)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < numNodeBC)
    {
        xyzdd[symm[i]] = Real_t(0.0);
    }
}

inline void ApplyAccelerationBoundaryConditionsForNodes(Domain* domain)
{
    Index_t dimBlock = 128;

    Index_t dimGrid = PAD_DIV(domain->numSymmX, dimBlock);
    if(domain->numSymmX > 0)
    {
// Alpaka Code
#ifdef ALPAKA
        using ApplyAccBoundaryConditionsNodes
            = lulesh_port_kernels::ApplyAccelerationBoundaryConditionsForNodes_kernel_class;
        ApplyAccBoundaryConditionsNodes ApplyAccBoundaryKernel(
            domain->numSymmX,
            domain->xdd.raw(),
            domain->symmX.raw());
        using Dim2 = alpaka::DimInt<2>;
        using Idx = std::size_t;
        using Vec2 = alpaka::Vec<Dim2, Idx>;
        alpaka_utils::alpakaExecuteBaseKernel<Dim2, Idx>(ApplyAccBoundaryKernel, Vec2{dimBlock, dimGrid}, false);
        // hipDeviceSynchronize();
#else
        // CUDA Code
        ApplyAccelerationBoundaryConditionsForNodes_kernel<<<dimGrid, dimBlock>>>(
            domain->numSymmX,
            domain->xdd.raw(),
            domain->symmX.raw());
#endif
    }

    dimGrid = PAD_DIV(domain->numSymmY, dimBlock);
    if(domain->numSymmY > 0)
    {
// Alpaka Code
#ifdef ALPAKA
        using ApplyAccBoundaryConditionsNodes
            = lulesh_port_kernels::ApplyAccelerationBoundaryConditionsForNodes_kernel_class;
        ApplyAccBoundaryConditionsNodes ApplyAccBoundaryKernel(
            domain->numSymmY,
            domain->ydd.raw(),
            domain->symmY.raw());
        using Dim2 = alpaka::DimInt<2>;
        using Idx = std::size_t;
        using Vec2 = alpaka::Vec<Dim2, Idx>;
        alpaka_utils::alpakaExecuteBaseKernel<Dim2, Idx>(ApplyAccBoundaryKernel, Vec2{dimBlock, dimGrid}, true);
        // hipDeviceSynchronize();
#else
        // CUDA Code
        ApplyAccelerationBoundaryConditionsForNodes_kernel<<<dimGrid, dimBlock>>>(
            domain->numSymmY,
            domain->ydd.raw(),
            domain->symmY.raw());
#endif
    }

    dimGrid = PAD_DIV(domain->numSymmZ, dimBlock);
    if(domain->numSymmZ > 0)
    {
// Alpaka Code
#ifdef ALPAKA
        using ApplyAccBoundaryConditionsNodes
            = lulesh_port_kernels::ApplyAccelerationBoundaryConditionsForNodes_kernel_class;
        ApplyAccBoundaryConditionsNodes ApplyAccBoundaryKernel(
            domain->numSymmZ,
            domain->zdd.raw(),
            domain->symmZ.raw());
        using Dim2 = alpaka::DimInt<2>;
        using Idx = std::size_t;
        using Vec2 = alpaka::Vec<Dim2, Idx>;
        alpaka_utils::alpakaExecuteBaseKernel<Dim2, Idx>(ApplyAccBoundaryKernel, Vec2{dimBlock, dimGrid}, true);

        // hipDeviceSynchronize();

// CUDA Code
#else
        ApplyAccelerationBoundaryConditionsForNodes_kernel<<<dimGrid, dimBlock>>>(
            domain->numSymmZ,
            domain->zdd.raw(),
            domain->symmZ.raw());
#endif
    }
}

__global__ void CalcPositionAndVelocityForNodes_kernel(
    int numNode,
    Real_t const deltatime,
    Real_t const u_cut,
    Real_t* __restrict__ x,
    Real_t* __restrict__ y,
    Real_t* __restrict__ z,
    Real_t* __restrict__ xd,
    Real_t* __restrict__ yd,
    Real_t* __restrict__ zd,
    Real_t const* __restrict__ xdd,
    Real_t const* __restrict__ ydd,
    Real_t const* __restrict__ zdd)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < numNode)
    {
        Real_t xdtmp, ydtmp, zdtmp, dt;
        dt = deltatime;

        xdtmp = xd[i] + xdd[i] * dt;
        ydtmp = yd[i] + ydd[i] * dt;
        zdtmp = zd[i] + zdd[i] * dt;

        if(FABS(xdtmp) < u_cut)
            xdtmp = 0.0;
        if(FABS(ydtmp) < u_cut)
            ydtmp = 0.0;
        if(FABS(zdtmp) < u_cut)
            zdtmp = 0.0;

        x[i] += xdtmp * dt;
        y[i] += ydtmp * dt;
        z[i] += zdtmp * dt;

        xd[i] = xdtmp;
        yd[i] = ydtmp;
        zd[i] = zdtmp;
    }
}

inline void CalcPositionAndVelocityForNodes(Real_t const u_cut, Domain* domain)
{
    Index_t dimBlock = 128;
    Index_t dimGrid = PAD_DIV(domain->numNode, dimBlock);
#ifdef ALPAKA
    // Alpaka Code
    using CalcPositionAndVelocityForNodes = lulesh_port_kernels::CalcPositionAndVelocityForNodes_kernel_class;
    CalcPositionAndVelocityForNodes CalcPosAndVeloKernel(
        domain->numNode,
        domain->deltatime_h,
        u_cut,
        domain->x.raw(),
        domain->y.raw(),
        domain->z.raw(),
        domain->xd.raw(),
        domain->yd.raw(),
        domain->zd.raw(),
        domain->xdd.raw(),
        domain->ydd.raw(),
        domain->zdd.raw());

    using Dim2 = alpaka::DimInt<2u>;
    using Idx = std::size_t;
    using Vec2 = alpaka::Vec<Dim2, Idx>;
    alpaka_utils::alpakaExecuteBaseKernel<Dim2, Idx>(CalcPosAndVeloKernel, Vec2{dimBlock, dimGrid}, true);
    // hipDeviceSynchronize();
#else
    // CUDA Code
    CalcPositionAndVelocityForNodes_kernel<<<dimGrid, dimBlock>>>(
        domain->numNode,
        domain->deltatime_h,
        u_cut,
        domain->x.raw(),
        domain->y.raw(),
        domain->z.raw(),
        domain->xd.raw(),
        domain->yd.raw(),
        domain->zd.raw(),
        domain->xdd.raw(),
        domain->ydd.raw(),
        domain->zdd.raw());
#endif

    // hipDeviceSynchronize();
    // cudaCheckError();
}

inline void LagrangeNodal(Domain* domain)
{
#ifdef SEDOV_SYNC_POS_VEL_EARLY
    Domain_member fieldData[6];
#endif

    Real_t u_cut = domain->u_cut;

    /* time of boundary condition evaluation is beginning of step for force and
     * acceleration boundary conditions. */
    CalcForceForNodes(domain);

#if USE_MPI
#    ifdef SEDOV_SYNC_POS_VEL_EARLY
    CommRecv(*domain, MSG_SYNC_POS_VEL, 6, domain->sizeX + 1, domain->sizeY + 1, domain->sizeZ + 1, false, false);
#    endif
#endif

    CalcAccelerationForNodes(domain);

    ApplyAccelerationBoundaryConditionsForNodes(domain);

    CalcPositionAndVelocityForNodes(u_cut, domain);
    // cudaCheckError();

#if USE_MPI
#    ifdef SEDOV_SYNC_POS_VEL_EARLY
    // initialize pointers
    domain->d_x = domain->x.raw();
    domain->d_y = domain->y.raw();
    domain->d_z = domain->z.raw();

    domain->d_xd = domain->xd.raw();
    domain->d_yd = domain->yd.raw();
    domain->d_zd = domain->zd.raw();

    fieldData[0] = &Domain::get_x;
    fieldData[1] = &Domain::get_y;
    fieldData[2] = &Domain::get_z;
    fieldData[3] = &Domain::get_xd;
    fieldData[4] = &Domain::get_yd;
    fieldData[5] = &Domain::get_zd;
printf(
  CommSendGpu(*domain, MSG_SYNC_POS_VEL, 6, fieldData,
           domain->sizeX + 1, domain->sizeY + 1, domain->sizeZ + 1,
           false, false, domain->streams[2]) ;
  CommSyncPosVelGpu(*domain, &domain->streams[2]) ;
#    endif
#endif

  return;
}

inline void CalcKinematicsAndMonotonicQGradient(Domain* domain)
{
    Index_t numElem = domain->numElem;
    Index_t padded_numElem = domain->padded_numElem;

    Index_t num_threads = numElem;

    Index_t const block_size = 64;
    Index_t dimGrid = PAD_DIV(num_threads, block_size);
#ifdef ALPAKA
    using CalcKinematicsAndMonotonicQGradient = lulesh_port_kernels::CalcKinematicsAndMonotonicQGradient_kernel_class;
    // cudaCheckError();
    CalcKinematicsAndMonotonicQGradient CalcKinematicsKernelObj(
        numElem,
        padded_numElem,
        domain->deltatime_h,
        domain->nodelist.raw(),
        domain->volo.raw(),
        domain->v.raw(),
        domain->x.raw(),
        domain->y.raw(),
        domain->z.raw(),
        domain->xd.raw(),
        domain->yd.raw(),
        domain->zd.raw(),
        domain->vnew->raw(),
        domain->delv.raw(),
        domain->arealg.raw(),
        domain->dxx->raw(),
        domain->dyy->raw(),
        domain->dzz->raw(),
        domain->vdov.raw(),
        domain->delx_zeta->raw(),
        domain->delv_zeta->raw(),
        domain->delx_xi->raw(),
        domain->delv_xi->raw(),
        domain->delx_eta->raw(),
        domain->delv_eta->raw(),
        domain->constraints_d.raw(),
        num_threads);

    using Dim2 = alpaka::DimInt<2>;
    using Idx = std::size_t;
    using Vec2 = alpaka::Vec<Dim2, Idx>;
    cudaCheckError();
    alpaka_utils::alpakaExecuteBaseKernel<Dim2, Idx>(CalcKinematicsKernelObj, Vec2{block_size, dimGrid}, true);
    // hipDeviceSynchronize();
    // cudaCheckError();
#else
    CalcKinematicsAndMonotonicQGradient_kernel<<<dimGrid, block_size>>>(
        numElem,
        padded_numElem,
        domain->deltatime_h,
        domain->nodelist.raw(),
        domain->volo.raw(),
        domain->v.raw(),
        domain->x.raw(),
        domain->y.raw(),
        domain->z.raw(),
        domain->xd.raw(),
        domain->yd.raw(),
        domain->zd.raw(),
        domain->vnew->raw(),
        domain->delv.raw(),
        domain->arealg.raw(),
        domain->dxx->raw(),
        domain->dyy->raw(),
        domain->dzz->raw(),
        domain->vdov.raw(),
        domain->delx_zeta->raw(),
        domain->delv_zeta->raw(),
        domain->delx_xi->raw(),
        domain->delv_xi->raw(),
        domain->delx_eta->raw(),
        domain->delv_eta->raw(),
        domain->bad_vol_h,
        num_threads);
    hipDeviceSynchronize();
    cudaCheckError();
#endif
}

inline void CalcMonotonicQRegionForElems(Domain* domain)
{
    Real_t const ptiny = Real_t(1.e-36);
    Real_t monoq_max_slope = domain->monoq_max_slope;
    Real_t monoq_limiter_mult = domain->monoq_limiter_mult;

    Real_t qlc_monoq = domain->qlc_monoq;
    Real_t qqc_monoq = domain->qqc_monoq;
    Index_t elength = domain->numElem;

    Index_t dimBlock = 128;
    Index_t dimGrid = PAD_DIV(elength, dimBlock);
#ifdef ALPAKA
    using CalcMonotonicQRegionForElems = lulesh_port_kernels::CalcMonotonicQRegionForElems_kernel_class;
    CalcMonotonicQRegionForElems CalcMonotonicQRegionKernel(
        qlc_monoq,
        qqc_monoq,
        monoq_limiter_mult,
        monoq_max_slope,
        ptiny,
        elength,
        domain->regElemlist.raw(),
        domain->elemBC.raw(),
        domain->lxim.raw(),
        domain->lxip.raw(),
        domain->letam.raw(),
        domain->letap.raw(),
        domain->lzetam.raw(),
        domain->lzetap.raw(),
        domain->delv_xi->raw(),
        domain->delv_eta->raw(),
        domain->delv_zeta->raw(),
        domain->delx_xi->raw(),
        domain->delx_eta->raw(),
        domain->delx_zeta->raw(),
        domain->vdov.raw(),
        domain->elemMass.raw(),
        domain->volo.raw(),
        domain->vnew->raw(),
        domain->qq.raw(),
        domain->ql.raw(),
        domain->q.raw(),
        domain->qstop,
        domain->constraints_d.raw());

    using Dim2 = alpaka::DimInt<2u>;
    using Idx = std::size_t;
    using Vec2 = alpaka::Vec<Dim2, Idx>;
    alpaka_utils::alpakaExecuteBaseKernel<Dim2, Idx>(CalcMonotonicQRegionKernel, Vec2{dimBlock, dimGrid}, true);
#else
    CalcMonotonicQRegionForElems_kernel<<<dimGrid, dimBlock>>>(
        qlc_monoq,
        qqc_monoq,
        monoq_limiter_mult,
        monoq_max_slope,
        ptiny,
        elength,
        domain->regElemlist.raw(),
        domain->elemBC.raw(),
        domain->lxim.raw(),
        domain->lxip.raw(),
        domain->letam.raw(),
        domain->letap.raw(),
        domain->lzetam.raw(),
        domain->lzetap.raw(),
        domain->delv_xi->raw(),
        domain->delv_eta->raw(),
        domain->delv_zeta->raw(),
        domain->delx_xi->raw(),
        domain->delx_eta->raw(),
        domain->delx_zeta->raw(),
        domain->vdov.raw(),
        domain->elemMass.raw(),
        domain->volo.raw(),
        domain->vnew->raw(),
        domain->qq.raw(),
        domain->ql.raw(),
        domain->q.raw(),
        domain->qstop,
        domain->bad_q_h);
#endif
    // hipDeviceSynchronize();
    // cudaCheckError();
}

std::vector<std::string> data;
int globalDataIndex = 0;

template<typename T>
void writeOut(T vec, std::string name)
{
    for(int i = 0; i < vec.size(); i++)
    {
        if(name=="delv"&&i==1000){
            std::cout<<vec[i]<<std::endl;
        }
        if(data[globalDataIndex] != std::to_string(vec[i]))
        {
            std::cout << "failure reading vec at " << i << " here " << std::to_string(vec[i])
                      << " lulesh:" << data[globalDataIndex] << "in file " << name << std::endl;
        }
        globalDataIndex++;
    }
}

void read_data()
{
    std::ifstream inputFile("/home/tim/Studium/Alpaka_Project/value_compare.txt");

    if(inputFile.is_open())
    {
        std::string value;
        while(inputFile >> value)
        {
            data.push_back(value);
        }
        inputFile.close();
        std::cout << "Data has been read from " << std::endl;
    }
    else
    {
        std::cerr << "Unable to open file: " << std::endl;
    }
    std::cout<<" size of Data "<<data.size()<<std::endl;
}

template<typename T>
void writeOutwriteOutWord(T word, std::string name)
{
    bool correct=true;
    if(data[globalDataIndex] != std::to_string(word))
    {
        std::cout << "failure reading word here" << std::to_string(word) << " lulesh: " << data[globalDataIndex]
                  << "in file " << name << std::endl;
        correct=false;
    }
    if(correct)std::cout<<" data correct for "<<name<<std::endl;

    globalDataIndex++;
}

template<typename T>
Vector_h<T> vector_h(Vector_d<T>& v)
{
    Vector_h<T> neu(v);
    neu = v;
    return std::move(neu);
}

void CheckErrorApply(

    Index_t length,
    Real_t rho0,
    Real_t e_cut,
    Real_t emin,
    Vector_d<Real_t>& ql,
    Vector_d<Real_t>& qq,
    Vector_d<Real_t>& vnew,
    Vector_d<Real_t>& v,
    Real_t pmin,
    Real_t p_cut,
    Real_t q_cut,
    Real_t eosvmin,
    Real_t eosvmax,
    Vector_d<Index_t>& regElemlist,
    //        const Index_t*  regElemlist,
    Vector_d<Real_t>& e,
    Vector_d<Real_t>& delv,
    Vector_d<Real_t>& p,
    Vector_d<Real_t>& q,
    Real_t ss4o3,
    Vector_d<Real_t>& ss,
    Real_t v_cut,
    Index_t bad_vol,
    Int_t const cost,
    Vector_d<Index_t>& regCSR,
    Vector_d<Index_t>& regReps,
    Index_t const numReg)
{
    writeOut(Vector_h<Real_t>(ql), "ql");
    writeOut(Vector_h<Real_t>(qq), "qq");
    writeOut(Vector_h<Real_t>(vnew), "vnew");
    writeOut(Vector_h<Real_t>(v), "v");
    writeOut(Vector_h<Index_t>(regElemlist), "regElemlist");
    writeOut(Vector_h<Real_t>(e), "e");
    writeOut(Vector_h<Real_t>(delv), "delv");
    writeOut(Vector_h<Real_t>(p), "p");
    writeOut(Vector_h<Real_t>(q), "q");
    writeOut(Vector_h<Real_t>(ss), "ss");
    writeOut(Vector_h<Index_t>(regCSR), "regCSR");
    writeOut(Vector_h<Index_t>(regReps), "regReps");
    writeOutwriteOutWord(length, "length");
    writeOutwriteOutWord(rho0, "rho0");
    writeOutwriteOutWord(e_cut, "e_cut");
    writeOutwriteOutWord(emin, "emin");
    writeOutwriteOutWord(pmin, "pmin");
    writeOutwriteOutWord(bad_vol, "bad_vol");
    writeOutwriteOutWord(p_cut, "p_cut");
    writeOutwriteOutWord(eosvmin, "eosvmin");
    writeOutwriteOutWord(eosvmax, "eosvmax");
    writeOutwriteOutWord(ss4o3, "ss4o3");
    writeOutwriteOutWord(v_cut, "v_cut");
    writeOutwriteOutWord(cost, "cost");
    writeOutwriteOutWord(numReg, "numReg");
}

void ApplyMaterialPropertiesAndUpdateVolume(Domain* domain)
{
    Index_t length = domain->numElem;
    static int iter = 0;
    if(length != 0)
    {
        #define ITER 2
        Index_t dimBlock = 128;
        Index_t dimGrid = PAD_DIV(length, dimBlock);
#define AlPAKA
#ifdef ALPAKA
        Vector_h constraints_h(domain->constraints_d);
            using ApplyMaterialPropertiesAndUpdateVolume
                = lulesh_port_kernels::ApplyMaterialPropertiesAndUpdateVolume_kernel_class;
            // cudaCheckError();
            ApplyMaterialPropertiesAndUpdateVolume ApplyMaterialPropertiesAndUpdateVolumeKernel;
            using Dim2 = alpaka::DimInt<2>;
            using Idx = std::size_t;
            using Vec2 = alpaka::Vec<Dim2, Idx>;
            if(iter==ITER){
                cudaCheckError();
                            read_data();
                CheckErrorApply(
                    length,
                    domain->refdens,
                    domain->e_cut,
                    domain->emin,
                    domain->ql, // dev
                    domain->qq, // dev
                    *domain->vnew, // dev,
                    domain->v, // dev,
                    domain->pmin,
                    domain->p_cut,
                    domain->q_cut,
                    domain->eosvmin,
                    domain->eosvmax,
                    domain->regElemlist, // dev,
                    domain->e, // dev,
                    domain->delv, // dev,
                    domain->p, // dev,
                    domain->q, // dev,
                    domain->ss4o3,
                    domain->ss, // dev,
                    domain->v_cut,
                    domain->constraints_h[2], // dev,
                    domain->cost,
                    domain->regCSR, // dev,
                    domain->regReps, // dev,
                    domain->numReg);
                cudaCheckError();
            }
            alpaka_utils::alpakaExecuteBaseKernel<Dim2, Idx>(
                ApplyMaterialPropertiesAndUpdateVolumeKernel,
                Vec2{dimBlock, dimGrid},
                true,
                length,
                domain->refdens,
                domain->e_cut,
                domain->emin,
                domain->ql.raw(),
                domain->qq.raw(),
                domain->vnew->raw(),
                domain->v.raw(), // error
                domain->pmin,
                domain->p_cut,
                domain->q_cut,
                domain->eosvmin,
                domain->eosvmax,
                domain->regElemlist.raw(),
                domain->e.raw(), // error
                domain->delv.raw(),
                domain->p.raw(), // error
                domain->q.raw(), // error
                domain->ss4o3,
                domain->ss.raw(), // error
                domain->v_cut,
                domain->constraints_d.raw(),
                domain->cost,
                domain->regCSR.raw(),
                domain->regReps.raw(),
                domain->numReg);
            constraints_h = domain->constraints_d;
            cudaCheckError();

            std::cout << std::endl;
            std::cout << " aft " << std::endl;

            std::cout<<" next iteration"<<std::endl;
            iter++;

#else

        ApplyMaterialPropertiesAndUpdateVolume_kernel<<<dimGrid, dimBlock>>>(
            length,
            domain->refdens,
            domain->e_cut,
            domain->emin,
            domain->ql.raw(),
            domain->qq.raw(),
            domain->vnew->raw(),
            domain->v.raw(),
            domain->pmin,
            domain->p_cut,
            domain->q_cut,
            domain->eosvmin,
            domain->eosvmax,
            domain->regElemlist.raw(),
            domain->e.raw(),
            domain->delv.raw(),
            domain->p.raw(),
            domain->q.raw(),
            domain->ss4o3,
            domain->ss.raw(),
            domain->v_cut,
            domain->bad_vol_h,
            domain->cost,
            domain->regCSR.raw(),
            domain->regReps.raw(),
            domain->numReg);
#endif
            // hipDeviceSynchronize();
            // cudaCheckError();
        }
    }

    inline void LagrangeElements(Domain * domain)
    {
        int allElem = domain->numElem + /* local elem */
                      2 * domain->sizeX * domain->sizeY + /* plane ghosts */
                      2 * domain->sizeX * domain->sizeZ + /* row ghosts */
                      2 * domain->sizeY * domain->sizeZ; /* col ghosts */

        domain->vnew = Allocator<Vector_d<Real_t>>::allocate(domain->numElem);
        domain->dxx = Allocator<Vector_d<Real_t>>::allocate(domain->numElem);
        domain->dyy = Allocator<Vector_d<Real_t>>::allocate(domain->numElem);
        domain->dzz = Allocator<Vector_d<Real_t>>::allocate(domain->numElem);

        domain->delx_xi = Allocator<Vector_d<Real_t>>::allocate(domain->numElem);
        domain->delx_eta = Allocator<Vector_d<Real_t>>::allocate(domain->numElem);
        domain->delx_zeta = Allocator<Vector_d<Real_t>>::allocate(domain->numElem);

        domain->delv_xi = Allocator<Vector_d<Real_t>>::allocate(allElem);
        domain->delv_eta = Allocator<Vector_d<Real_t>>::allocate(allElem);
        domain->delv_zeta = Allocator<Vector_d<Real_t>>::allocate(allElem);

#if USE_MPI
        CommRecv(*domain, MSG_MONOQ, 3, domain->sizeX, domain->sizeY, domain->sizeZ, true, true);
#endif

        /*********************************************/
        /*  Calc Kinematics and Monotic Q Gradient   */
        /*********************************************/
        CalcKinematicsAndMonotonicQGradient(domain);

#if USE_MPI
        Domain_member fieldData[3];

        // initialize pointers
        domain->d_delv_xi = domain->delv_xi->raw();
        domain->d_delv_eta = domain->delv_eta->raw();
        domain->d_delv_zeta = domain->delv_zeta->raw();

        fieldData[0] = &Domain::get_delv_xi;
        fieldData[1] = &Domain::get_delv_eta;
        fieldData[2] = &Domain::get_delv_zeta;

        CommSendGpu(
            *domain,
            MSG_MONOQ,
            3,
            fieldData,
            domain->sizeX,
            domain->sizeY,
            domain->sizeZ,
            true,
            true,
            domain->streams[2]);
        CommMonoQGpu(*domain, domain->streams[2]);
#endif

        Allocator<Vector_d<Real_t>>::free(domain->dxx, domain->numElem);
        Allocator<Vector_d<Real_t>>::free(domain->dyy, domain->numElem);
        Allocator<Vector_d<Real_t>>::free(domain->dzz, domain->numElem);

        /**********************************
         *    Calc Monotic Q Region
         **********************************/
        CalcMonotonicQRegionForElems(domain);

        Allocator<Vector_d<Real_t>>::free(domain->delx_xi, domain->numElem);
        Allocator<Vector_d<Real_t>>::free(domain->delx_eta, domain->numElem);
        Allocator<Vector_d<Real_t>>::free(domain->delx_zeta, domain->numElem);

        Allocator<Vector_d<Real_t>>::free(domain->delv_xi, allElem);
        Allocator<Vector_d<Real_t>>::free(domain->delv_eta, allElem);
        Allocator<Vector_d<Real_t>>::free(domain->delv_zeta, allElem);

        ApplyMaterialPropertiesAndUpdateVolume(domain);
        Allocator<Vector_d<Real_t>>::free(domain->vnew, domain->numElem);
    }

    template<int block_size>
    __global__
#ifdef DOUBLE_PRECISION
        __launch_bounds__(128, 16)
#else
    __launch_bounds__(128, 16)
#endif
            void
            CalcTimeConstraintsForElems_kernel(
                Index_t length,
                Real_t qqc2,
                Real_t dvovmax,
                Index_t * matElemlist,
                Real_t * ss,
                Real_t * vdov,
                Real_t * arealg,
                Real_t * dev_mindtcourant,
                Real_t * dev_mindthydro)
    {
        int tid = threadIdx.x;
        int i = blockDim.x * blockIdx.x + tid;

        __shared__ volatile Real_t s_mindthydro[block_size];
        __shared__ volatile Real_t s_mindtcourant[block_size];

        Real_t mindthydro = Real_t(1.0e+20);
        Real_t mindtcourant = Real_t(1.0e+20);

        Real_t dthydro = mindthydro;
        Real_t dtcourant = mindtcourant;

        while(i < length)
        {
            Index_t indx = matElemlist[i];
            Real_t vdov_tmp = vdov[indx];

            // Computing dt_hydro
            if(vdov_tmp != Real_t(0.))
            {
                Real_t dtdvov = dvovmax / (FABS(vdov_tmp) + Real_t(1.e-20));
                if(dthydro > dtdvov)
                {
                    dthydro = dtdvov;
                }
            }
            if(dthydro < mindthydro)
                mindthydro = dthydro;

            // Computing dt_courant
            Real_t ss_tmp = ss[indx];
            Real_t area_tmp = arealg[indx];
            Real_t dtf = ss_tmp * ss_tmp;

            dtf += ((vdov_tmp < 0.) ? qqc2 * area_tmp * area_tmp * vdov_tmp * vdov_tmp : 0.);

            dtf = area_tmp / SQRT(dtf);

            /* determine minimum timestep with its corresponding elem */
            if(vdov_tmp != Real_t(0.) && dtf < dtcourant)
            {
                dtcourant = dtf;
            }

            if(dtcourant < mindtcourant)
                mindtcourant = dtcourant;

            i += gridDim.x * blockDim.x;
        }

        s_mindthydro[tid] = mindthydro;
        s_mindtcourant[tid] = mindtcourant;

        __syncthreads();

        // Do shared memory reduction
        if(block_size >= 1024)
        {
            if(tid < 512)
            {
                s_mindthydro[tid] = min(s_mindthydro[tid], s_mindthydro[tid + 512]);
                s_mindtcourant[tid] = min(s_mindtcourant[tid], s_mindtcourant[tid + 512]);
            }
            __syncthreads();
        }

        if(block_size >= 512)
        {
            if(tid < 256)
            {
                s_mindthydro[tid] = min(s_mindthydro[tid], s_mindthydro[tid + 256]);
                s_mindtcourant[tid] = min(s_mindtcourant[tid], s_mindtcourant[tid + 256]);
            }
            __syncthreads();
        }

        if(block_size >= 256)
        {
            if(tid < 128)
            {
                s_mindthydro[tid] = min(s_mindthydro[tid], s_mindthydro[tid + 128]);
                s_mindtcourant[tid] = min(s_mindtcourant[tid], s_mindtcourant[tid + 128]);
            }
            __syncthreads();
        }

        if(block_size >= 128)
        {
            if(tid < 64)
            {
                s_mindthydro[tid] = min(s_mindthydro[tid], s_mindthydro[tid + 64]);
                s_mindtcourant[tid] = min(s_mindtcourant[tid], s_mindtcourant[tid + 64]);
            }
            __syncthreads();
        }

        if(tid < 32)
        {
            s_mindthydro[tid] = min(s_mindthydro[tid], s_mindthydro[tid + 32]);
            s_mindtcourant[tid] = min(s_mindtcourant[tid], s_mindtcourant[tid + 32]);
        }

        if(tid < 16)
        {
            s_mindthydro[tid] = min(s_mindthydro[tid], s_mindthydro[tid + 16]);
            s_mindtcourant[tid] = min(s_mindtcourant[tid], s_mindtcourant[tid + 16]);
        }
        if(tid < 8)
        {
            s_mindthydro[tid] = min(s_mindthydro[tid], s_mindthydro[tid + 8]);
            s_mindtcourant[tid] = min(s_mindtcourant[tid], s_mindtcourant[tid + 8]);
        }
        if(tid < 4)
        {
            s_mindthydro[tid] = min(s_mindthydro[tid], s_mindthydro[tid + 4]);
            s_mindtcourant[tid] = min(s_mindtcourant[tid], s_mindtcourant[tid + 4]);
        }
        if(tid < 2)
        {
            s_mindthydro[tid] = min(s_mindthydro[tid], s_mindthydro[tid + 2]);
            s_mindtcourant[tid] = min(s_mindtcourant[tid], s_mindtcourant[tid + 2]);
        }
        if(tid < 1)
        {
            s_mindthydro[tid] = min(s_mindthydro[tid], s_mindthydro[tid + 1]);
            s_mindtcourant[tid] = min(s_mindtcourant[tid], s_mindtcourant[tid + 1]);
        }

        // Store in global memory
        if(tid == 0)
        {
            dev_mindtcourant[blockIdx.x] = s_mindtcourant[0];
            dev_mindthydro[blockIdx.x] = s_mindthydro[0];
        }
    }

    template<int block_size>
    __global__ void CalcMinDtOneBlock(
        Real_t * dev_mindthydro,
        Real_t * dev_mindtcourant,
        Real_t * dtcourant,
        Real_t * dthydro,
        Index_t shared_array_size)
    {
        __shared__ volatile Real_t s_data[block_size];
        int tid = threadIdx.x;

        if(blockIdx.x == 0)
        {
            if(tid < shared_array_size)
                s_data[tid] = dev_mindtcourant[tid];
            else
                s_data[tid] = 1.0e20;

            __syncthreads();

            if(block_size >= 1024)
            {
                if(tid < 512)
                {
                    s_data[tid] = min(s_data[tid], s_data[tid + 512]);
                }
                __syncthreads();
            }
            if(block_size >= 512)
            {
                if(tid < 256)
                {
                    s_data[tid] = min(s_data[tid], s_data[tid + 256]);
                }
                __syncthreads();
            }
            if(block_size >= 256)
            {
                if(tid < 128)
                {
                    s_data[tid] = min(s_data[tid], s_data[tid + 128]);
                }
                __syncthreads();
            }
            if(block_size >= 128)
            {
                if(tid < 64)
                {
                    s_data[tid] = min(s_data[tid], s_data[tid + 64]);
                }
                __syncthreads();
            }
            if(tid < 32)
            {
                s_data[tid] = min(s_data[tid], s_data[tid + 32]);
            }
            if(tid < 16)
            {
                s_data[tid] = min(s_data[tid], s_data[tid + 16]);
            }
            if(tid < 8)
            {
                s_data[tid] = min(s_data[tid], s_data[tid + 8]);
            }
            if(tid < 4)
            {
                s_data[tid] = min(s_data[tid], s_data[tid + 4]);
            }
            if(tid < 2)
            {
                s_data[tid] = min(s_data[tid], s_data[tid + 2]);
            }
            if(tid < 1)
            {
                s_data[tid] = min(s_data[tid], s_data[tid + 1]);
            }

            if(tid < 1)
            {
                *(dtcourant) = s_data[0];
            }
        }
        else if(blockIdx.x == 1)
        {
            if(tid < shared_array_size)
                s_data[tid] = dev_mindthydro[tid];
            else
                s_data[tid] = 1.0e20;

            __syncthreads();

            if(block_size >= 1024)
            {
                if(tid < 512)
                {
                    s_data[tid] = min(s_data[tid], s_data[tid + 512]);
                }
                __syncthreads();
            }
            if(block_size >= 512)
            {
                if(tid < 256)
                {
                    s_data[tid] = min(s_data[tid], s_data[tid + 256]);
                }
                __syncthreads();
            }
            if(block_size >= 256)
            {
                if(tid < 128)
                {
                    s_data[tid] = min(s_data[tid], s_data[tid + 128]);
                }
                __syncthreads();
            }
            if(block_size >= 128)
            {
                if(tid < 64)
                {
                    s_data[tid] = min(s_data[tid], s_data[tid + 64]);
                }
                __syncthreads();
            }
            if(tid < 32)
            {
                s_data[tid] = min(s_data[tid], s_data[tid + 32]);
            }
            if(tid < 16)
            {
                s_data[tid] = min(s_data[tid], s_data[tid + 16]);
            }
            if(tid < 8)
            {
                s_data[tid] = min(s_data[tid], s_data[tid + 8]);
            }
            if(tid < 4)
            {
                s_data[tid] = min(s_data[tid], s_data[tid + 4]);
            }
            if(tid < 2)
            {
                s_data[tid] = min(s_data[tid], s_data[tid + 2]);
            }
            if(tid < 1)
            {
                s_data[tid] = min(s_data[tid], s_data[tid + 1]);
            }

            if(tid < 1)
            {
                *(dthydro) = s_data[0];
            }
        }
    }

    inline void CalcTimeConstraintsForElems(Domain * domain)
    {
        Real_t qqc = domain->qqc;
        Real_t qqc2 = Real_t(64.0) * qqc * qqc;
        Real_t dvovmax = domain->dvovmax;

        Index_t const length = domain->numElem;

        int const max_dimGrid = 1024;
        int const dimBlock = 128;
        int dimGrid = std::min(max_dimGrid, PAD_DIV(length, dimBlock));

        Vector_d<Real_t>* dev_mindtcourant = Allocator<Vector_d<Real_t>>::allocate(dimGrid);
        Vector_d<Real_t>* dev_mindthydro = Allocator<Vector_d<Real_t>>::allocate(dimGrid);
        // hipDeviceSynchronize();
#ifdef ALPAKA
        using CalcTimeConstraintsForElems = lulesh_port_kernels::CalcTimeConstraintsForElems_kernel_class<dimBlock>;
        // cudaCheckError();
        CalcTimeConstraintsForElems CalcTimeConstraintsKernel(
            length,
            qqc2,
            dvovmax,
            domain->matElemlist.raw(),
            domain->ss.raw(),
            domain->vdov.raw(),
            domain->arealg.raw(),
            dev_mindtcourant->raw(),
            dev_mindthydro->raw());

        using Dim2 = alpaka::DimInt<2>;
        using Idx = std::size_t;
        using Vec2 = alpaka::Vec<Dim2, Idx>;

        alpaka_utils::alpakaExecuteBaseKernel<Dim2, Idx>(CalcTimeConstraintsKernel, Vec2{dimBlock, dimGrid}, true);

        // hipDeviceSynchronize();
        // cudaCheckError();

        // TODO: CalcMinDtOneBlock
        using CalcMinDtOneBlock = lulesh_port_kernels::CalcMinDtOneBlock_class<max_dimGrid>;
        // cudaCheckError();
        CalcMinDtOneBlock CalcMinDtOneBlockKernel(
            dev_mindthydro->raw(),
            dev_mindtcourant->raw(),
            domain->constraints_d.raw(),
            dimGrid);

        using Dim2 = alpaka::DimInt<2>;
        using Idx = std::size_t;
        using Vec2 = alpaka::Vec<Dim2, Idx>;

        alpaka_utils::alpakaExecuteBaseKernel<Dim2, Idx>(
            CalcMinDtOneBlockKernel,
            Vec2{max_dimGrid, 2},
            true); // Should be started with two blocks!

        // hipDeviceSynchronize();
        // cudaCheckError();

#else
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(CalcTimeConstraintsForElems_kernel<dimBlock>), hipFuncCachePreferShared);

    CalcTimeConstraintsForElems_kernel<dimBlock><<<dimGrid, dimBlock>>>(
        length,
        qqc2,
        dvovmax,
        domain->matElemlist.raw(),
        domain->ss.raw(),
        domain->vdov.raw(),
        domain->arealg.raw(),
        dev_mindtcourant->raw(),
        dev_mindthydro->raw());

    // TODO: if dimGrid < 1024, should launch less threads
    CalcMinDtOneBlock<max_dimGrid><<<2, max_dimGrid, max_dimGrid * sizeof(Real_t), domain->streams[1]>>>(
        dev_mindthydro->raw(),
        dev_mindtcourant->raw(),
        domain->dtcourant_h,
        domain->dthydro_h,
        dimGrid);
#endif

        // hipEventRecord(domain->time_constraint_computed,domain->streams[1]);

        Allocator<Vector_d<Real_t>>::free(dev_mindtcourant, dimGrid);
        Allocator<Vector_d<Real_t>>::free(dev_mindthydro, dimGrid);
    }

    inline void LagrangeLeapFrog(Domain * domain)
    {
        /* calculate nodal forces, accelerations, velocities, positions, with
         * applied boundary conditions and slide surface considerations */
        LagrangeNodal(domain);

        /* calculate element quantities (i.e. velocity gradient & q), and update
         * material states */
        LagrangeElements(domain);

        CalcTimeConstraintsForElems(domain);
    }

    void printUsage(char* argv[])
    {
        printf("Usage: \n");
        printf("Unstructured grid:  %s -u <file.lmesh> \n", argv[0]);
        printf("Structured grid:    %s -s numEdgeElems \n", argv[0]);
        printf("\nExamples:\n");
        printf("%s -s 45\n", argv[0]);
        printf("%s -u sedov15oct.lmesh\n", argv[0]);
    }

#ifdef SAMI

#    ifdef __cplusplus
    extern "C"
    {
#    endif
#    include "silo.h"
#    ifdef __cplusplus
    }
#    endif

#    define MAX_LEN_SAMI_HEADER 10

#    define SAMI_HDR_NUMBRICK 0
#    define SAMI_HDR_NUMNODES 3
#    define SAMI_HDR_NUMMATERIAL 4
#    define SAMI_HDR_INDEX_START 6
#    define SAMI_HDR_MESHDIM 7

#    define MAX_ADJACENCY 14 /* must be 14 or greater */

    void DumpSAMI(Domain * domain, char* name)
    {
        DBfile* fp;
        int headerLen = MAX_LEN_SAMI_HEADER;
        int headerInfo[MAX_LEN_SAMI_HEADER];
        char varName[] = "brick_nd0";
        char coordName[] = "x";
        int version = 121;
        int numElem = int(domain->numElem);
        int numNode = int(domain->numNode);
        int count;

        int* materialID;
        int* nodeConnect;
        double* nodeCoord;

        if((fp = DBCreate(name, DB_CLOBBER, DB_LOCAL, NULL, DB_PDB)) == NULL)
        {
            printf("Couldn't create file %s\n", name);
            exit(1);
        }

        for(int i = 0; i < MAX_LEN_SAMI_HEADER; ++i)
        {
            headerInfo[i] = 0;
        }
        headerInfo[SAMI_HDR_NUMBRICK] = numElem;
        headerInfo[SAMI_HDR_NUMNODES] = numNode;
        headerInfo[SAMI_HDR_NUMMATERIAL] = 1;
        headerInfo[SAMI_HDR_INDEX_START] = 1;
        headerInfo[SAMI_HDR_MESHDIM] = 3;

        DBWrite(fp, "mesh_data", headerInfo, &headerLen, 1, DB_INT);

        count = 1;
        DBWrite(fp, "version", &version, &count, 1, DB_INT);

        nodeConnect = new int[numElem];

        Vector_h<Index_t> nodelist_h = domain->nodelist;

        for(Index_t i = 0; i < 8; ++i)
        {
            for(Index_t j = 0; j < numElem; ++j)
            {
                nodeConnect[j] = int(nodelist_h[i * domain->padded_numElem + j]) + 1;
            }
            varName[8] = '0' + i;
            DBWrite(fp, varName, nodeConnect, &numElem, 1, DB_INT);
        }

        delete[] nodeConnect;

        nodeCoord = new double[numNode];

        Vector_h<Real_t> x_h = domain->x;
        Vector_h<Real_t> y_h = domain->y;
        Vector_h<Real_t> z_h = domain->z;

        for(Index_t i = 0; i < 3; ++i)
        {
            for(Index_t j = 0; j < numNode; ++j)
            {
                Real_t coordVal;
                switch(i)
                {
                case 0:
                    coordVal = double(x_h[j]);
                    break;
                case 1:
                    coordVal = double(y_h[j]);
                    break;
                case 2:
                    coordVal = double(z_h[j]);
                    break;
                }
                nodeCoord[j] = coordVal;
            }
            coordName[0] = 'x' + i;
            DBWrite(fp, coordName, nodeCoord, &numNode, 1, DB_DOUBLE);
        }

        delete[] nodeCoord;

        materialID = new int[numElem];

        for(Index_t i = 0; i < numElem; ++i)
            materialID[i] = 1;

        DBWrite(fp, "brick_material", materialID, &numElem, 1, DB_INT);

        delete[] materialID;

        DBClose(fp);
    }
#endif

#ifdef SAMI
    void DumpDomain(Domain * domain)
    {
        char meshName[64];
        printf("Dumping SAMI file\n");
        sprintf(meshName, "sedov_%d.sami", int(domain->cycle));

        DumpSAMI(domain, meshName);
    }
#endif

    void write_solution(Domain * locDom)
    {
        Vector_h<Real_t> x_h = locDom->x;
        Vector_h<Real_t> y_h = locDom->y;
        Vector_h<Real_t> z_h = locDom->z;

        std::stringstream filename;
        filename << "xyz.asc";

        FILE* fout = fopen(filename.str().c_str(), "wb");

        for(Index_t i = 0; i < locDom->numNode; i++)
        {
            fprintf(fout, "%10d\n", i);
            fprintf(fout, "%.10f\n", x_h[i]);
            fprintf(fout, "%.10f\n", y_h[i]);
            fprintf(fout, "%.10f\n", z_h[i]);
        }
        fclose(fout);
    }

    ///////////////////////////////////////////////////////////////////////////
    void InitMeshDecomp(Int_t numRanks, Int_t myRank, Int_t * col, Int_t * row, Int_t * plane, Int_t * side)
    {
        Int_t testProcs;
        Int_t dx, dy, dz;
        Int_t myDom;

        // Assume cube processor layout for now
        testProcs = Int_t(cbrt(Real_t(numRanks)) + 0.5);
        if(testProcs * testProcs * testProcs != numRanks)
        {
            printf("Num processors must be a cube of an integer (1, 8, 27, ...)\n");
#if USE_MPI
            MPI_Abort(MPI_COMM_WORLD, -1);
#else
        exit(-1);
#endif
        }
        if(sizeof(Real_t) != 4 && sizeof(Real_t) != 8)
        {
            printf("MPI operations only support float and double right now...\n");
#if USE_MPI
            MPI_Abort(MPI_COMM_WORLD, -1);
#else
        exit(-1);
#endif
        }
        if(MAX_FIELDS_PER_MPI_COMM > CACHE_COHERENCE_PAD_REAL)
        {
            printf("corner element comm buffers too small.  Fix code.\n");
#if USE_MPI
            MPI_Abort(MPI_COMM_WORLD, -1);
#else
        exit(-1);
#endif
        }

        dx = testProcs;
        dy = testProcs;
        dz = testProcs;

        // temporary test
        if(dx * dy * dz != numRanks)
        {
            printf("error -- must have as many domains as procs\n");
#if USE_MPI
            MPI_Abort(MPI_COMM_WORLD, -1);
#else
        exit(-1);
#endif
        }
        Int_t remainder = dx * dy * dz % numRanks;
        if(myRank < remainder)
        {
            myDom = myRank * (1 + (dx * dy * dz / numRanks));
        }
        else
        {
            myDom = remainder * (1 + (dx * dy * dz / numRanks)) + (myRank - remainder) * (dx * dy * dz / numRanks);
        }

        *col = myDom % dx;
        *row = (myDom / dx) % dy;
        *plane = myDom / (dx * dy);
        *side = testProcs;

        return;
    }

    void VerifyAndWriteFinalOutput(
        Real_t elapsed_time,
        Domain & locDom,
        Int_t its,
        Int_t nx,
        Int_t numRanks,
        bool structured)
    {
        size_t free_mem, total_mem, used_mem;
        hipMemGetInfo(&free_mem, &total_mem);
        used_mem = total_mem - free_mem;
#if LULESH_SHOW_PROGRESS == 0
        printf("   Used Memory         =  %8.4f Mb\n", used_mem / (1024. * 1024.));
#endif

        // GrindTime1 only takes a single domain into account, and is thus a good way
        // to measure processor speed indepdendent of MPI parallelism. GrindTime2
        // takes into account speedups from MPI parallelism
        Real_t grindTime1;
        Real_t grindTime2;
        if(structured)
        {
            grindTime1 = ((elapsed_time * 1e6) / its) / (nx * nx * nx);
            grindTime2 = ((elapsed_time * 1e6) / its) / (nx * nx * nx * numRanks);
        }
        else
        {
            grindTime1 = ((elapsed_time * 1e6) / its) / (locDom.numElem);
            grindTime2 = ((elapsed_time * 1e6) / its) / (locDom.numElem * numRanks);
        }
        // Copy Energy back to Host
        std::cout << structured << std::endl;
        if(structured)
        {
            Real_t e_zero;
            // Real_t* d_ezero_ptr = locDom.e.raw() + locDom.octantCorner; /* octant
            // corner supposed to be 0 */
            Vector_h e_all(locDom.e);
            e_zero = e_all[locDom.octantCorner];
            // hipMemcpy(&e_zero, d_ezero_ptr, sizeof(Real_t), hipMemcpyDeviceToHost);

            printf("Run completed:  \n");
            printf("   Problem size        =  %i \n", nx);
            printf("   MPI tasks           =  %i \n", numRanks);
            printf("   Iteration count     =  %i \n", its);
            printf("   Final Origin Energy = %12.6e \n", e_zero);

            Real_t MaxAbsDiff = Real_t(0.0);
            Real_t TotalAbsDiff = Real_t(0.0);
            Real_t MaxRelDiff = Real_t(0.0);
            for(Index_t j = 0; j < nx; ++j)
            {
                for(Index_t k = j + 1; k < nx; ++k)
                {
                    Real_t AbsDiff = FABS(e_all[j * nx + k] - e_all[k * nx + j]);
                    TotalAbsDiff += AbsDiff;

                    if(MaxAbsDiff < AbsDiff)
                        MaxAbsDiff = AbsDiff;

                    Real_t RelDiff = AbsDiff / e_all[k * nx + j];

                    if(MaxRelDiff < RelDiff)
                        MaxRelDiff = RelDiff;
                }
            }

            // Quick symmetry check
            printf("   Testing Plane 0 of Energy Array on rank 0:\n");
            printf("        MaxAbsDiff   = %12.6e\n", MaxAbsDiff);
            printf("        TotalAbsDiff = %12.6e\n", TotalAbsDiff);
            printf("        MaxRelDiff   = %12.6e\n\n", MaxRelDiff);
        }

        // Timing information
        printf("\nElapsed time         = %10.2f (s)\n", elapsed_time);
        printf("Grind time (us/z/c)  = %10.8g (per dom)  (%10.8g overall)\n", grindTime1, grindTime2);
        printf("FOM                  = %10.8g (z/s)\n\n",
               1000.0 / grindTime2); // zones per second

        bool write_solution_flag = true;
        if(write_solution_flag)
        {
            write_solution(&locDom);
        }

        return;
    }

    int main(int argc, char* argv[])
    {
        if(argc < 3)
        {
            printUsage(argv);
            exit(LFileError);
        }
        if(strcmp(argv[1], "-u") != 0 && strcmp(argv[1], "-s") != 0)
        {
            printUsage(argv);
            exit(LFileError);
        }
        int num_iters = -1;
        if(argc == 5)
        {
            num_iters = atoi(argv[4]);
        }

        bool structured = (strcmp(argv[1], "-s") == 0);
        Int_t numRanks;
        Int_t myRank;

#ifdef TEST

        if(test::test_main())
        {
            std::cout << " Some Tests failed << ABORTING LULESH >> " << std::endl;
            return 1;
        }
#endif
#if USE_MPI
        Domain_member fieldData;

        MPI_Init(&argc, &argv);
        MPI_Comm_size(MPI_COMM_WORLD, &numRanks);
        MPI_Comm_rank(MPI_COMM_WORLD, &myRank);
#else
    numRanks = 1;
    myRank = 0;
#endif

        /* assume cube subdomain geometry for now */
        Index_t nx = atoi(argv[2]);

        Domain* locDom;

        // Set up the mesh and decompose. Assumes regular cubes for now
        Int_t col, row, plane, side;
        using std::cout;
        using std::endl;

        InitMeshDecomp(numRanks, myRank, &col, &row, &plane, &side);

        // TODO: change default nr to 11
        Int_t nr = 11;
        Int_t balance = 1;
        Int_t cost = 1;

        // TODO: modify this constructor to account for new fields
        // TODO: setup communication buffers
        locDom = NewDomain(argv, numRanks, col, row, plane, nx, side, structured, nr, balance, cost);
#if USE_MPI
        // copy to the host for mpi transfer
        locDom->h_nodalMass = locDom->nodalMass;

        fieldData = &Domain::get_nodalMass;

        // Initial domain boundary communication
        CommRecv(*locDom, MSG_COMM_SBN, 1, locDom->sizeX + 1, locDom->sizeY + 1, locDom->sizeZ + 1, true, false);
        CommSend(
            *locDom,
            MSG_COMM_SBN,
            1,
            &fieldData,
            locDom->sizeX + 1,
            locDom->sizeY + 1,
            locDom->sizeZ + 1,
            true,
            false);
        CommSBN(*locDom, 1, &fieldData);

        // copy back to the device
        locDom->nodalMass = locDom->h_nodalMass;

        // End initialization
        MPI_Barrier(MPI_COMM_WORLD);
#endif

        // timestep to solution
        int its = 0;

        if(myRank == 0)
        {
            if(structured)
                printf("Running until t=%f, Problem size=%dx%dx%d\n", locDom->stoptime, nx, nx, nx);
            else
                printf("Running until t=%f, Problem size=%d \n", locDom->stoptime, locDom->numElem);
        }

        hipProfilerStart();

#if USE_MPI
        double start = MPI_Wtime();
#else
    timeval start;
    gettimeofday(&start, NULL);
#endif

        while(true)
        {
            // this has been moved after computation of volume forces to hide launch
            // latencies
            // TimeIncrement(locDom) ;

            LagrangeLeapFrog(locDom);

            checkErrors(locDom, its, myRank);

#if LULESH_SHOW_PROGRESS
            if(myRank == 0)
                printf("cycle = %d, time = %e, dt=%e\n", its + 1, double(locDom->time_h), double(locDom->deltatime_h));
#endif
            its++;
            if(its == num_iters)
                break;
        }
        // make sure GPU finished its work
        // hipDeviceSynchronize();
        // Use reduced max elapsed time
        double elapsed_time;
#if USE_MPI
        elapsed_time = MPI_Wtime() - start;
#else
    timeval end;
    gettimeofday(&end, NULL);
    elapsed_time = (double) (end.tv_sec - start.tv_sec) + ((double) (end.tv_usec - start.tv_usec)) / 1'000'000;
#endif

        double elapsed_timeG;
#if USE_MPI
        MPI_Reduce(&elapsed_time, &elapsed_timeG, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);
#else
    elapsed_timeG = elapsed_time;
#endif

        hipProfilerStop();

        if(myRank == 0)
            VerifyAndWriteFinalOutput(elapsed_timeG, *locDom, its, nx, numRanks, structured);

#ifdef SAMI
        DumpDomain(locDom);
#endif

#if USE_MPI
        MPI_Finalize();
#endif

        return 0;
    }
